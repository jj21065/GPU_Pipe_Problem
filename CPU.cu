#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include<windows.h>

#define N 29
#define Pr 100.0
float rs0 = 1;

float r01 = 1;
float r16 = 5;
float r03 = 5;
float r12 = 1;
float r34 = 3;

float r24 = 1;
float r25 = 1;
float r56 = 3;
float r47 = 1;
float r3e = 1;

float r57 = 1;
float r78 = 2;
float r69 = 3;
float r89 = 3;

float r9EA = 1;

float r8a = 1;
float rab = 3;
float rad = 2;
float r9g = 1;

float reb = 2;
float rdg = 5;
float rbc = 2;
float rdc = 1;

float ref = 5;
float rcf = 2;
float rgh = 1;
float rfh = 2;
float rhEB = 1;


void MatrixVectorProduct(float *a, float*p, float *ap, int n);

float Dot(float *v1, float *v2, int n);

void Sum_Store(float *store, float *x, float scalar, float*v, int n);

void conjgrad(float*A, float*x, float*b);

void Newton_Raphson_method(float *x, float*r);

void Compute_R(float*x, float*r, float *R);

void Compute_J(float*x, float*r, int n);

void Compute_invJ_mul_R(float*x, float*r, float*R, float*invJ_R);

void inverse(float*A, float* invA, int n);

float Hardy_Cross_method(float* Q,float*r);

int main()
{
	float* Q = new float[N];
	float* r = new float[N];
	// pipe parameter 
	float init_Q = sqrt(Pr);
	float Q01 = init_Q / 2.0;
	float Q24 = init_Q / 8.0;
	float Q57 = init_Q / 8.0;
	float Q16 = init_Q / 4.0;
	float Q25 = init_Q / 8.0;
	float Q78 = init_Q / 4.0;
	float Q12 = init_Q / 4.0;
	float Q56 = 0;
	float Q89 = 0;
	float Q03 = init_Q / 2.0;
	float Q3e = init_Q / 2.0;
	float Q69 = init_Q / 4.0;
	float Q34 = 0;
	float Q47 = init_Q / 8.0;
	float Q8a = init_Q / 4.0;
	float Qab = init_Q / 8.0;
	float Qad = init_Q / 8.0;
	float Qeb = 0;
	float Qbc = init_Q / 8.0;
	float Qdc = 0;
	float Qdg = init_Q / 8.0;
	float Q9g = 0;
	float Qgh = init_Q / 8.0;
	float Qef = init_Q / 2.0;
	float Qcf = init_Q / 8.0;
	float Qfh = init_Q * 5.0 / 8.0;
	float QS0 = init_Q;
	float QEA = init_Q / 4.0;
	float QEB = init_Q * 3.0 / 4.0;


	Q[0] = Q01;
	Q[1] = Q24;
	Q[2] = Q57;
	Q[3] = Q16;
	Q[4] = Q25;
	Q[5] = Q78;
	Q[6] = Q12;
	Q[7] = Q56;
	Q[8] = Q89;
	Q[9] = Q03;
	Q[10] = Q3e;
	Q[11] = Q69;
	Q[12] = Q34;
	Q[13] = Q47;
	Q[14] = Q8a;
	Q[15] = Qab;
	Q[16] = Qad;
	Q[17] = Qeb;
	Q[18] = Qbc;
	Q[19] = Qdc;
	Q[20] = Qdg;
	Q[21] = Q9g;
	Q[22] = Qgh;
	Q[23] = Qef;
	Q[24] = Qcf;
	Q[25] = Qfh;
	Q[26] = QS0;
	Q[27] = QEA;
	Q[28] = QEB;

	r[0] = r01;
	r[1] = r24;
	r[2] = r57;
	r[3] = r16;
	r[4] = r25;
	r[5] = r78;
	r[6] = r12;
	r[7] = r56;
	r[8] = r89;
	r[9] = r03;
	r[10] = r3e;
	r[11] = r69;
	r[12] = r34;
	r[13] = r47;
	r[14] = r8a;
	r[15] = rab;
	r[16] = rad;
	r[17] = reb;
	r[18] = rbc;
	r[19] = rdc;
	r[20] = rdg;
	r[21] = r9g;
	r[22] = rgh;
	r[23] = ref;
	r[24] = rcf;
	r[25] = rfh;
	r[26] = rs0;
	r[27] = r9EA;
	r[28] = rhEB;

	// initial conditions 

	int iter_no =2000;
	int i;
	float Error = 1e5;

	SYSTEMTIME t1, t2;
	GetLocalTime(&t1);

	for (i = 0; i < iter_no; i++)
	{
		Error = Hardy_Cross_method(Q,r);
		/*if (Error < 1e-8)
			break;*/
		//Newton_Raphson_method(Q, r);
	}
	GetLocalTime(&t2);
	float time = t2.wSecond - t1.wSecond + (t2.wMilliseconds - t1.wMilliseconds) / 1000.0;
	printf("iter : %d, time : %g\n", i, time);
	for (int i = 0; i < 29; i++)
		printf("final Q[%d] = %g\n", i, Q[i]);

	/*
		printf("final Q[0] = %g\n", Q[0]);
		printf("final Q9-EA = %g\n", Q[27]);
		printf("final Qh-EB = %g\n", Q[28]);*/
	system("pause");

	delete[] Q;
	delete[] r;

	return 0;
}

float Hardy_Cross_method(float* Q,float *r)
{


	float R[11] = { 0 };

	R[0] = -(r[0] * Q[0] * abs(Q[0]) + r[6] * Q[6] * abs(Q[6]) + r[1] * Q[1] * abs(Q[1]) - r[12] * Q[12] * abs(Q[12]) - r[9] * Q[9] * abs(Q[9]))
		/ (2 * r[0] * fabs(Q[0]) + 2 * r[6] * fabs(Q[6]) + 2 * r[1] * fabs(Q[1]) + 2 * r[12] * fabs(Q[12]) + 2 * r[9] * fabs(Q[9]));

	R[1] = -(r[3] * Q[3] * abs(Q[3]) - r[7] * Q[7] * abs(Q[7]) - r[4] * Q[4] * abs(Q[4]) - r[6] * Q[6] * abs(Q[6]))
		/ (2 * r[3] * fabs(Q[3]) + 2 * r[7] * fabs(Q[7]) + 2 * r[4] * fabs(Q[4]) + 2 * r[6] * fabs(Q[6]));

	R[2] = -(r[4] * Q[4] * abs(Q[4]) + r[2] * Q[2] * abs(Q[2]) - r[13] * Q[13] * abs(Q[13]) - r[1] * Q[1] * abs(Q[1]))
		/ (2 * r[4] * fabs(Q[4]) + 2 * r[2] * fabs(Q[2]) + 2 * r[13] * fabs(Q[13]) + 2 * r[1] * fabs(Q[1]));

	R[3] = -(r[12] * Q[12] * abs(Q[12]) + r[13] * Q[13] * abs(Q[13]) + r[5] * Q[5] * abs(Q[5]) + r[14] * Q[14] * abs(Q[14]) + r[15] * Q[15] * abs(Q[15]) - r[17] * Q[17] * abs(Q[17]) - r[10] * Q[10] * abs(Q[10]))
		/ (2 * r[12] * fabs(Q[12]) + 2 * r[13] * fabs(Q[13]) + 2 * r[5] * fabs(Q[5]) + 2 * r[14] * fabs(Q[14]) + 2 * r[15] * fabs(Q[15]) + 2 * r[17] * fabs(Q[17]) + 2 * r[10] * fabs(Q[10]));

	R[4] = -(r[7] * Q[7] * abs(Q[7]) + r[11] * Q[11] * abs(Q[11]) - r[8] * Q[8] * abs(Q[8]) - r[5] * Q[5] * abs(Q[5]) - r[2] * Q[2] * abs(Q[2]))
		/ (2 * r[7] * fabs(Q[7]) + 2 * r[11] * fabs(Q[11]) + 2 * r[8] * fabs(Q[8]) + 2 * r[5] * fabs(Q[5]) + 2 * r[2] * fabs(Q[2]));

	R[5] = -(r[8] * Q[8] * abs(Q[8]) - r[21] * Q[21] * abs(Q[21]) - r[20] * Q[20] * abs(Q[20]) - r[16] * Q[16] * abs(Q[16]) - r[14] * Q[14] * abs(Q[14]))
		/ (2 * r[8] * fabs(Q[8]) + 2 * r[21] * fabs(Q[21]) + 2 * r[20] * fabs(Q[20]) + 2 * r[16] * fabs(Q[16]) + 2 * r[14] * fabs(Q[14]));

	R[6] = -(r[16] * Q[16] * abs(Q[16]) + r[19] * Q[19] * abs(Q[19]) - r[18] * Q[18] * abs(Q[18]) - r[15] * Q[15] * abs(Q[15]))
		/ (2 * r[16] * fabs(Q[16]) + 2 * r[19] * fabs(Q[19]) + 2 * r[18] * fabs(Q[18]) + 2 * r[15] * fabs(Q[15]));

	R[7] = -(r[17] * Q[17] * abs(Q[17]) + r[18] * Q[18] * abs(Q[18]) + r[24] * Q[24] * abs(Q[24]) - r[23] * Q[23] * abs(Q[23]))
		/ (2 * r[17] * fabs(Q[17]) + 2 * r[18] * fabs(Q[18]) + 2 * r[24] * fabs(Q[24]) + 2 * r[23] * fabs(Q[23]));

	R[8] = -(r[20] * Q[20] * abs(Q[20]) + r[22] * Q[22] * abs(Q[22]) - r[25] * Q[25] * abs(Q[25]) - r[24] * Q[24] * abs(Q[24]) - r[19] * Q[19] * abs(Q[19]))
		/ (2 * r[20] * fabs(Q[20]) + 2 * r[22] * fabs(Q[22]) + 2 * r[25] * fabs(Q[25]) + 2 * r[24] * fabs(Q[24]) + 2 * r[19] * fabs(Q[19]));

	R[9] = -(-Pr + r[26] * Q[26] * abs(Q[26]) + r[0] * Q[0] * abs(Q[0]) + r[3] * Q[3] * abs(Q[3]) + r[11] * Q[11] * abs(Q[11]) + r[27] * Q[27] * abs(Q[27]))
		/ (2 * r[26] * fabs(Q[26]) + 2 * r[0] * fabs(Q[0]) + 2 * r[3] * fabs(Q[3]) + 2 * r[11] * fabs(Q[11]) + 2 * r[27] * fabs(Q[27]));

	R[10] = -(-Pr + r[26] * Q[26] * abs(Q[26]) + r[9] * Q[9] * abs(Q[9]) + r[10] * Q[10] * abs(Q[10]) + r[23] * Q[23] * abs(Q[23]) + r[25] * Q[25] * abs(Q[25]) + r[28] * Q[28] * abs(Q[28]))
		/ (2 * r[26] * fabs(Q[26]) + 2 * r[9] * fabs(Q[9]) + 2 * r[10] * fabs(Q[10]) + 2 * r[23] * fabs(Q[23]) + 2 * r[25] * fabs(Q[25]) + 2 * r[28] * fabs(Q[28]));


	Q[0] = Q[0] + R[0] + R[9];
	Q[1] = Q[1] + R[0] - R[2];
	Q[2] = Q[2] + R[2] - R[4];
	Q[3] = Q[3] + R[1] + R[9];
	Q[4] = Q[4] - R[1] + R[2];
	Q[5] = Q[5] + R[3] - R[4];
	Q[6] = Q[6] + R[0] - R[1];
	Q[7] = Q[7] - R[1] + R[4];
	Q[8] = Q[8] - R[4] + R[5];
	Q[9] = Q[9] - R[0] + R[10];
	Q[10] = Q[10] - R[3] + R[10];
	Q[11] = Q[11] + R[4] + R[9];
	Q[12] = Q[12] - R[0] + R[3];
	Q[13] = Q[13] - R[2] + R[3];
	Q[14] = Q[14] + R[3] - R[5];
	Q[15] = Q[15] + R[3] - R[6];
	Q[16] = Q[16] + R[6] - R[5];
	Q[17] = Q[17] - R[3] + R[7];
	Q[18] = Q[18] - R[6] + R[7];
	Q[19] = Q[19] + R[6] - R[8];
	Q[20] = Q[20] - R[5] + R[8];
	Q[21] = Q[21] - R[5];
	Q[22] = Q[22] + R[8];
	Q[23] = Q[23] - R[7] + R[10];
	Q[24] = Q[24] + R[7] - R[8];
	Q[25] = Q[25] - R[8] + R[10];
	Q[26] = Q[26] + R[9] + R[10];
	Q[27] = Q[27] + R[9];
	Q[28] = Q[28] + R[10];

	float tmpErr = 0;
	for (int i = 0; i < 11; i++)
	{
		tmpErr = tmpErr + R[i] * R[i];
	}
	return tmpErr;

}

void Newton_Raphson_method(float *x, float*r)
{

	/// Computet R 
	float invJR[N] = { 0 };
	float R[N] = { 0 };
	Compute_R(x, r, R);
	/// Compute J 
	Compute_invJ_mul_R(x, r, R, invJR);

	for (int i = 0; i < N; i++)
	{
		x[i] = x[i] - invJR[i];
	}
	// x = x - inv(J')*R'
}

void Compute_R(float*x, float*r, float *R)
{


	R[0] = (r01*x[1] * abs(x[1]) + r12*x[4] * abs(x[4]) + r24*x[6] * abs(x[6]) - r34*x[5] * abs(x[5]) - r03*x[3] * abs(x[3]));

	R[1] = (r16*x[2] * abs(x[2]) - r56*x[8] * abs(x[8]) - r25*x[7] * abs(x[7]) - r12*x[4] * abs(x[4]));

	R[2] = (r25*x[7] * abs(x[7]) + r57*x[10] * abs(x[10]) - r47*x[9] * abs(x[9]) - r24*x[6] * abs(x[6]));

	R[3] = (r34*x[5] * abs(x[5]) + r47*x[9] * abs(x[9]) + r78*x[11] * abs(x[11]) + r8a*x[16] * abs(x[16]) + rab*x[17] * abs(x[17]) - reb*x[20] * abs(x[20]) - r3e*x[15] * abs(x[15]));

	R[4] = (r56*x[8] * abs(x[8]) + r69*x[12] * abs(x[12]) - r89*x[13] * abs(x[13]) - r78*x[11] * abs(x[11]) - r57*x[10] * abs(x[10]));

	R[5] = (r89*x[13] * abs(x[13]) - r9g*x[19] * abs(x[19]) - rdg*x[21] * abs(x[21]) - rad*x[18] * abs(x[18]) - r8a*x[16] * abs(x[16]));

	R[6] = (rad*x[18] * abs(x[18]) + rdc*x[23] * abs(x[23]) - rbc*x[22] * abs(x[22]) - rab*x[17] * abs(x[17]));

	R[7] = (reb*x[20] * abs(x[20]) + rbc*x[22] * abs(x[22]) + rcf*x[25] * abs(x[25]) - ref*x[24] * abs(x[24]));

	R[8] = (rdg*x[21] * abs(x[21]) + rgh*x[26] * abs(x[26]) - rfh*x[27] * abs(x[27]) - rcf*x[25] * abs(x[25]) - rdc*x[23] * abs(x[23]));

	R[9] = x[25] + x[24] - x[27];

	R[10] = (rs0*x[0] * abs(x[0]) + r01*x[1] * abs(x[1]) + r16*x[2] * abs(x[2]) + r69*x[12] * abs(x[12]) + r9EA*x[14] * abs(x[14]) - 10);

	R[11] = (rs0*x[0] * abs(x[0]) + r03*x[3] * abs(x[3]) + r3e*x[15] * abs(x[15]) + ref*x[24] * abs(x[24]) + rfh*x[27] * abs(x[27]) + rhEB*x[28] * abs(x[28]) - 10);

	R[12] = x[0] - x[1] - x[3];

	R[13] = x[1] - x[4] - x[2];

	R[14] = x[4] - x[6] - x[7];

	R[15] = x[3] - x[5] - x[15];

	R[16] = x[5] + x[6] - x[9];

	R[17] = x[7] - x[8] - x[10];

	R[18] = x[2] + x[8] - x[12];

	R[19] = x[10] + x[9] - x[11];

	R[20] = x[11] - x[13] - x[16];

	R[21] = x[13] + x[12] + x[19] - x[14];

	R[22] = x[16] - x[17] - x[18];

	R[23] = x[15] - x[20] - x[24];

	R[24] = x[20] + x[17] - x[22];

	R[25] = x[18] - x[21] - x[23];

	R[26] = x[21] + x[19] - x[26];

	R[27] = x[26] + x[27] - x[28];

	R[28] = x[22] + x[23] - x[25];


}

void Compute_invJ_mul_R(float*x, float*r, float*R, float*invJ_R)
{
	/// Conjugate gradient method 
	float A[N*N] = { 0 };
	
	A[0] = 0; A[1] = 2 * r[1] * x[1];  A[4] = 2 * r[4] * x[4]; A[6] = 2 * r[6] * x[6]; A[5] = -2 * r[5] * x[5]; A[3] = -2 * r[3] * x[3];

	A[29] = 0; A[29 + 2] = 2 * r[2] * x[2]; A[29 + 8] = -2 * r[8] * x[8]; A[29 + 7] = -2 * r[7] * x[7]; A[29 + 4] = -2 * r[4] * x[4];

	A[58] = 0; A[58 + 7] = 2 * r[7] * x[7]; A[58 + 10] = 2 * r[10] * x[10]; A[58 + 9] = -2 * r[9] * x[9]; A[58 + 6] = -2 * r[6] * x[6];

	A[87] = 0; A[87 + 5] = 2 * r[5] * x[5]; A[9] = 2 * r[9] * x[9]; A[87 + 11] = 2 * r[11] * x[11]; A[87 + 16] = 2 * r[16] * x[16]; A[87 + 17] = 2 * r[17] * x[17]; A[87 + 20] = -2 * r[20] * x[20]; A[87 + 15] = -2 * r[15] * x[15];

	A[116] = 0; A[116 + 8] = 2 * r[8] * x[8]; A[116 + 12] = 2 * r[12] * x[12]; A[116 + 13] = -2 * r[13] * x[13]; A[116 + 11] = -2 * r[11] * x[11]; A[116 + 10] = -2 * r[10] * x[10];
	
	///
	
	A[145] = 0; A[145 + 13] = 2 * r[13] * x[13]; A[145 + 19] = -2 * r[19] * x[19]; A[145 + 21] = -2 * r[21] * x[21]; A[145 + 18] = -2 * r[18] * x[18]; A[145 + 16] = -2 * r[16] * x[16];

	A[174] = 0; A[174 + 18] = 2 * r[18] * x[18]; A[174 + 23] = 2 * r[23] * x[23]; A[174 + 22] = -2 * r[22] * x[22]; A[174 + 17] = -2 * r[17] * x[17];

	A[203] = 0; A[203 + 20] = 2 * r[20] * x[20]; A[203 + 22] = 2 * r[22] * x[22]; A[203 + 25] = 2 * r[25] * x[25]; A[203 + 24] = -2 * r[24] * x[24];

	A[232] = 0; A[232 + 21] = 2 * r[21] * x[21]; A[232 + 26] = 2 * r[26] * x[26]; A[232 + 27] = -2 * r[27] * x[27]; A[232 + 25] = -2 * r[25] * x[25]; A[232 + 23] = -2 * r[23] * x[23];

	A[261] = 0; A[261 + 25] = 1; A[261 + 24] = 1; A[261 + 27] = -1;

	///

	A[290] = 2 * r[0] * x[0]; A[290 + 1] = 2 * r[1] * x[1]; A[290 + 2] = 2 * r[2] * x[2]; A[290 + 12] = 2 * r[12] * x[12]; A[290 + 14] = 2 * r[14] * x[14];

	A[319] = 2 * r[0] * x[0]; A[319 + 3] = 2 * r[3] * x[3]; A[319 + 15] = 2 * r[15] * x[15]; A[319 + 24] = 2 * r[24] * x[24]; A[319 + 27] = 2 * r[27] * x[27]; A[319 + 28] = 2 * r[28] * x[28];

	A[348] = 1; A[348 + 1] = -1; A[348 + 3] = -1;

	A[377] = 0; A[377 + 1] = 1; A[377 + 4] = -1; A[377 + 2] = 1;

	A[406] = 0; A[406+4] = 1; A[406+6] = -1; A[406+7] = -1;

	A[435] = 0; A[435+3] = 1; A[435+5] = -1; A[435+15] = -1;

	A[464] = 0; A[464+5] = 1; A[464+6] = 1; A[464+9] = -1;

	A[493] = 0; A[493+7] = 1; A[493+8] = 1; A[493+10] = -1;

	A[522] = 0; A[522+2] = 1; A[522+8] = 1; A[522+12] = -1;

	A[551] = 0; A[551+10] = 1; A[551+9] = 1; A[551+11] = -1;

	A[580] = 0; A[580+11] = 1; A[580+13] = -1; A[580+16] = -1;

	A[609] = 0; A[609+13] = 1; A[609+12] = 1; A[609+19] = 1; A[609+14] = -1;

	A[638] = 0; A[638+16] = 1; A[638+17] = -1; A[638+18] = -1;

	A[667] = 0; A[667+15] = 1; A[667+20] = -1; A[667+24] = -1;

	A[696] = 0; A[696+20] = 1; A[696+17] = 1; A[696+22] = -1;

	A[725] = 0; A[725+18] = 1; A[725+21] = -1; A[725+23] = -1;

	A[754] = 0; A[754+21] = 1; A[754+19] = 1; A[754+26] = -1;

	A[783] = 0; A[783+26] = 1; A[783+27] = 1; A[783+28] = -1;

	A[812] = 0; A[812+22] = 1; A[812+23] = 1; A[812+25] = -1;

	FILE*pfile;
	pfile = fopen("output.txt", "w");

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			fprintf(pfile, "%g\t", A[i*N + j]);
		}
		fprintf(pfile, "\n");
	}
	fclose(pfile);
	//	conjgrad(A, invJ_R, R);
	float Atemp[N*N] = { 0 };
	//inverse(A, Atemp, N);
	conjgrad(A, invJ_R, R);
	//MatrixVectorProduct(Atemp, R, invJ_R, N);

}

void MatrixVectorProduct(float *a, float*p, float *ap, int n)
{
	int i = 0;
	int j = 0;

	for (i = 0; i < n; i++){
		ap[i] = 0.0;
		for (j = 0; j < n; j++){
			ap[i] = ap[i] + a[i*n + j] * p[j];
		}
	}
}

float Dot(float *v1, float *v2, int n)
{
	int i;
	float ans = 0.0;
	for (i = 0; i < n; i++){
		ans += v1[i] * v2[i];

	}
	return ans;
}

void Sum_Store(float *store, float *x, float scalar, float*v, int n)
{
	int i = 0;
	for (i = 0; i < n; i++){
		store[i] = x[i] + scalar * v[i];
	}
}

void conjgrad(float*A, float*x, float*b)
{
	float ap[N];
	float p[N];
	float rr[N];
	float rsold = 0;
	float rsnew = 0;
	int i = 0;
	MatrixVectorProduct(A, x, ap, N);
	for (i = 0; i < N; i++){
		rr[i] = b[i] - ap[i];
		p[i] = rr[i];
	}
	rsold = Dot(rr, rr, N);

	for (i = 0; i < N; i++){
		MatrixVectorProduct(A, p, ap, N);
		/*	for (int j = 0; j < N*N; j++){
				printf("ap[%d] = %g\n", j, ap[j]);
				}*/
		float tmpap = Dot(ap, p, N);
		float alpha = rsold / (tmpap);
		Sum_Store(x, x, alpha, p, N);
		Sum_Store(rr, rr, -alpha, ap, N);
		rsnew = Dot(rr, rr, N);
		if (sqrt(rsnew) < 1e-10)
			break;
		Sum_Store(p, rr, (rsnew / rsold), p, N);
		rsold = rsnew;

	}
	// 	for (i = 0; i < N; i++)
	// 		printf("x[%d] = %g\n", i, x[i]);


}

void inverse(float*A, float* invA, int n)
{
	int i = 0, j = 0, k = 0;
	float d;
	float *a = new float[n*n * 2];

	for (i = 0; i < n*n * 2; i++)
		a[i] = 0;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			a[i*n * 2 + j] = A[i*n + j];
		}
	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	for (i = 0; i < n; i++)
		for (j = 0; j < 2 * n; j++)
			if (j == (i + n))
				a[i*n * 2 + j] = 1;

	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	/************** partial pivoting **************/
	for (i = n - 1; i > 0; i--)
	{
		if (a[(i - 1)*n * 2 + 0] < a[i*n * 2 + 0])
			for (j = 0; j < n * 2; j++)
			{
				d = a[i*n * 2 + j];
				a[i*n * 2 + j] = a[(i - 1)*n * 2 + j];
				a[(i - 1)*n * 2 + j] = d;
			}
	}
	//cout << "pivoted output: " << endl;
	/*for (i = 0; i < n; i++)
	{
	for (j = 0; j < 2 * n; j++)
	{
	printf("%g\t", a[i*n*2 + j]);
	}
	printf("\n");
	}*/
	/********** reducing to diagonal  matrix ***********/

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
			if (j != i)
			{
				d = a[j*n * 2 + i] / a[i*n * 2 + i];
				for (k = 0; k < n * 2; k++)
					a[j*n * 2 + k] -= a[i*n * 2 + k] * d;
			}
	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n * 2 + j]);
	//	}
	//	printf("\n");
	//}
	/************** reducing to unit matrix *************/
	for (i = 0; i < n; i++)
	{
		d = a[i*n * 2 + i];
		if (d != 0)
			for (j = 0; j < n * 2; j++)
				a[i*n * 2 + j] = a[i*n * 2 + j] / d;
	}

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
			invA[i*n + j] = a[i*n * 2 + n + j];

	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j <  n; j++)
	//	{
	//		printf("%g\t", invA[i*n+j]);
	//	}
	//	printf("\n");
	//}
	delete[] a;
}

float norm(float *a, int n)
{

	float value = 0;
	for (int i = 0; i < n; i++)
	{
		value = value + a[i] * a[i];
	}
	return value;
}

//void BICG(float *A, float *x, float *b)
//{
//	int iter = N;
//	int flag = 0;
//
//	float norm2 = norm(b, N);
//	if (norm2 == 0.0)
//		norm2 = 1.0;
//	float ap[N] = { 0 };
//	float r[N] = { 0 };
//	MatrixVectorProduct(A, x, ap, N);
//	for (int i = 0; i < N; i++){
//		r[i] = b[i] - ap[i];
//	}
//		
//	float error = norm(r, N) / norm2;
//	if (error < 0.001) 
//		return;
//
//	float r_tld[N];
//	for (int i = 0; i < N; i++)
//		r_tld[i] = r[i];
//
//	for(int i = 0;i<iter;i++)
//		                   
//
//		z = M \ r;
//	z_tld = M' \ r_tld;
//		rho = (z'*r_tld );
//		if (rho == 0.0),
//			break
//			end
//
//			if (iter > 1), % direction vectors
//				beta = rho / rho_1;
//	p = z + beta*p;
//	p_tld = z_tld + beta*p_tld;
//			else
//				p = z;
//	p_tld = z_tld;
//	end
//
//		q = A*p;                            % compute residual pair
//		q_tld = A'*p_tld;
//		alpha = rho / (p_tld'*q );
//
//		x = x + alpha*p;                    % update approximation
//		r = r - alpha*q;
//	r_tld = r_tld - alpha*q_tld;
//
//	error = norm(r) / bnrm2;          % check convergence
//		if (error <= tol), break, end
//
//			rho_1 = rho;
//
//	end
//
//		if (error <= tol), % converged
//			flag = 0;
//	elseif(rho == 0.0), % breakdown
//		flag = -1;
//		else
//			flag = 1;                           % no convergence
//			end
//
//}