#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<windows.h>
#define N 11
#define NN 29
#define TPB 29
#define BPG (N+TPB-1)/TPB
float *Q; // 29
float *d_Q; // 29
float *dR;// 11
void Allocate_Memory(int n)
{
	size_t size = NN*sizeof(float);
	
	Q = (float*)malloc(size);
	hipError_t error = hipMalloc((void**)&d_Q, size);
	printf("Allocate mem dQ: %s\n", hipGetErrorString(error));
	error = hipMalloc((void**)&dR, N*sizeof(float));
	printf("Allocate mem dR : %s\n", hipGetErrorString(error));
}

void Free_Memory()
{
	if (Q)
		free(Q);
	hipError_t error = hipFree(d_Q);
	printf("Free mem dQ : %s\n", hipGetErrorString(error));
	error = hipFree(dR);
	printf("Free mem dR: %s\n", hipGetErrorString(error));
}

void CopyMemToDevice(int n)
{
	/*for (int i = 0; i < n; i++)
	{
	R[i] = data[i];
	}*/
	size_t size = NN*sizeof(float);
	hipError_t error = hipMemcpy(d_Q, Q, size, hipMemcpyHostToDevice);
	printf("Memcpy Host to Device : %s\n", hipGetErrorString(error));
}

void CopyMemToHost( int n)
{
	size_t size = NN*sizeof(float);
	hipError_t error = hipMemcpy(Q, d_Q, size, hipMemcpyDeviceToHost);
	printf("Memcpy Device to Host : %s\n", hipGetErrorString(error));
	/*for (int i = 0; i < n; i++)
	{
	data[i] = R[i];
	}*/
}

__global__ void Compute_Q(float* pQ, float*dR, int n);
__global__ void Add_Q(float* pQ, float*dR, int n);
int main()
{
	
	
	Allocate_Memory(N);
	hipMemset(dR, 0.0,N*sizeof(float));

	float rs0 = 1;
	Q[0] = sqrt(10.0 / rs0);
	Q[1] = Q[0];
	Q[2] = Q[0];
	Q[3] = 0;
	Q[4] = 0;
	Q[5] = 0;
	Q[6] = 0;
	Q[7] = 0;
	Q[8] = 0;
	Q[9] = 0;
	Q[10] = 0;
	Q[11] = 0;
	Q[12] = 0;
	Q[13] = 0;
	Q[14] = Q[0];
	Q[15] = 0;
	Q[16] = 0;
	Q[17] = 0;
	Q[18] = 0;
	Q[19] = 0;
	Q[20] = 0;
	Q[21] = 0;
	Q[22] = 0;
	Q[23] = 0;
	Q[24] = 0;
	Q[25] = 0;
	Q[26] = 0;
	Q[27] = Q[0];
	Q[28] = 0;

	int n = 2;
	int iter_no = 5000;
	int i;

	printf("inital Q[0] = %g\n", Q[0]);

	LARGE_INTEGER startTime, endTime, fre;
	double times;
	QueryPerformanceFrequency(&fre); //���oCPU�W�v
	QueryPerformanceCounter(&startTime); //���o�}����{�b�g�L�X��CPU Cycle

	CopyMemToDevice(NN);
	for (i = 0; i < iter_no; i++)
	{
		Compute_Q<<<BPG,TPB>>>(d_Q,dR, N);
		Add_Q << <1, 1 >> >(d_Q, dR, N);
		
	}
	CopyMemToHost(NN);

	QueryPerformanceCounter(&endTime); //���o�}����{�����槹���g�L�X��CPU Cycle
	times = ((double)endTime.QuadPart - (double)startTime.QuadPart) / fre.QuadPart;
	

	printf("time consume : %f\n", times);
	printf("final Q[0] = %g\n", Q[0]);
	printf("final Q9-EA = %g\n", Q[27]);
	printf("final Qh-EB = %g\n", Q[28]);

	system("pause");
	Free_Memory();
	return 0;
}

__global__ void Compute_Q(float* pQ,float*dR,int n )
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	// pipe parameter 
	float r01 = 1;
	float r16 = 5;
	float r12 = 1;
	float r03 = 5;
	float r34 = 3;

	float r24 = 1;
	float r25 = 1;
	float r56 = 3;
	float r3e = 1;
	float r47 = 1;

	float r57 = 1;
	float r78 = 2;
	float r89 = 3;
	float r69 = 3;

	float r8a = 1;
	float reb = 2;
	float rdg = 5;
	float ref = 5;

	float rab = 3;
	float rbc = 2;
	float r9g = 1;
	float rcf = 2;

	float rad = 2;
	float rdc = 1;
	float rgh = 1;
	float rfh = 2;

	float rs0 = 1;
	float r9EA = 1;
	float rhEB = 1;

	//float R[11] = { 0 };

	if (i < n)
	{
	
		// calculate adjust flow pQ
		if (i == 0)
		{
			if (2 * (r01*abs(pQ[1]) + r12*abs(pQ[3]) + r24*abs(pQ[6]) + r34*abs(pQ[5]) + r03*abs(pQ[4])) > 0)
				dR[0] = -(r01*pQ[1] * abs(pQ[1]) + r12*pQ[3] * abs(pQ[3]) + r24*pQ[6] * abs(pQ[6]) - r34*pQ[5] * abs(pQ[5]) - r03*pQ[4] * abs(pQ[4])) / (2 * (r01*abs(pQ[1]) + r12*abs(pQ[3]) + r24*abs(pQ[6]) + r34*abs(pQ[5]) + r03*abs(pQ[4])));
		}
		else if (i == 1)
		{
			if (2 * (r16*abs(pQ[2]) + r56*abs(pQ[8]) + r25*abs(pQ[7]) + r12*abs(pQ[3])) > 0)
				dR[1] = -(r16*pQ[2] * abs(pQ[2]) - r56*pQ[8] * abs(pQ[8]) - r25*pQ[7] * abs(pQ[7]) - r12*pQ[3] * abs(pQ[3])) / (2 * (r16*abs(pQ[2]) + r56*abs(pQ[8]) + r25*abs(pQ[7]) + r12*abs(pQ[3])));
		}
		else if (i == 2){
			if (2 * (r34*abs(pQ[5]) + r47*abs(pQ[10]) + r78*abs(pQ[12]) + r8a*abs(pQ[15]) + rab*abs(pQ[19]) + reb*abs(pQ[16]) + r3e*abs(pQ[9])) > 0)
				dR[2] = -(r34*pQ[5] * abs(pQ[5]) + r47*pQ[10] * abs(pQ[10]) + r78*pQ[12] * abs(pQ[12]) + r8a*pQ[15] * abs(pQ[15]) + rab*pQ[19] * abs(pQ[19]) - reb*pQ[16] * abs(pQ[16]) - r3e*pQ[9] * abs(pQ[9])) / (2 * (r34*abs(pQ[5]) + r47*abs(pQ[10]) + r78*abs(pQ[12]) + r8a*abs(pQ[15]) + rab*abs(pQ[19]) + reb*abs(pQ[16]) + r3e*abs(pQ[9])));
		}
		else if (i == 3){
			if (2 * (r56*abs(pQ[8]) + r69*abs(pQ[14]) + r89*abs(pQ[13]) + r78*abs(pQ[12]) + r57*abs(pQ[11])) > 0)
				dR[3] = -(r56*pQ[8] * abs(pQ[8]) + r69*pQ[14] * abs(pQ[14]) - r89*pQ[13] * abs(pQ[13]) - r78*pQ[12] * abs(pQ[12]) - r57*pQ[11] * abs(pQ[11])) / (2 * (r56*abs(pQ[8]) + r69*abs(pQ[14]) + r89*abs(pQ[13]) + r78*abs(pQ[12]) + r57*abs(pQ[11])));
		}
		else if (i == 4){
			if (2 * (r89*abs(pQ[13]) + r9g*abs(pQ[21]) + rdg*abs(pQ[17]) + rad*abs(pQ[23]) + r8a*abs(pQ[15])) > 0)
				dR[4] = -(r89*pQ[13] * abs(pQ[13]) - r9g*pQ[21] * abs(pQ[21]) - rdg*pQ[17] * abs(pQ[17]) - rad*pQ[23] * abs(pQ[23]) - r8a*pQ[15] * abs(pQ[15])) / (2 * (r89*abs(pQ[13]) + r9g*abs(pQ[21]) + rdg*abs(pQ[17]) + rad*abs(pQ[23]) + r8a*abs(pQ[15])));
		}
		else if (i == 5){
			if (2 * (rad*abs(pQ[23]) + rdc*abs(pQ[24]) + rbc*abs(pQ[20]) + rab*abs(pQ[19])) > 0)
				dR[5] = -(rad*pQ[23] * abs(pQ[23]) + rdc*pQ[24] * abs(pQ[24]) - rbc*pQ[20] * abs(pQ[20]) - rab*pQ[19] * abs(pQ[19])) / (2 * (rad*abs(pQ[23]) + rdc*abs(pQ[24]) + rbc*abs(pQ[20]) + rab*abs(pQ[19])));
		}
		else if (i == 6){
			if (2 * (reb*abs(pQ[16]) + rbc*abs(pQ[20]) + rcf*abs(pQ[22]) + ref*abs(pQ[18])) > 0)
				dR[6] = -(reb*pQ[16] * abs(pQ[16]) + rbc*pQ[20] * abs(pQ[20]) + rcf*pQ[22] * abs(pQ[22]) - ref*pQ[18] * abs(pQ[18])) / (2 * (reb*abs(pQ[16]) + rbc*abs(pQ[20]) + rcf*abs(pQ[22]) + ref*abs(pQ[18])));
		}
		else if (i == 7){
			if (2 * (rdg*abs(pQ[17]) + rgh*abs(pQ[25]) + rfh*abs(pQ[26]) + rcf*abs(pQ[22]) + rdc*abs(pQ[24])) > 0)
				dR[7] = -(rdg*pQ[17] * abs(pQ[17]) + rgh*pQ[25] * abs(pQ[25]) - rfh*pQ[26] * abs(pQ[26]) - rcf*pQ[22] * abs(pQ[22]) - rdc*pQ[24] * abs(pQ[24])) / (2 * (rdg*abs(pQ[17]) + rgh*abs(pQ[25]) + rfh*abs(pQ[26]) + rcf*abs(pQ[22]) + rdc*abs(pQ[24])));
		}
		else if (i == 8)
		{
			if (2 * (r25*abs(pQ[7]) + r57*abs(pQ[11]) + r47*abs(pQ[10]) + r24*abs(pQ[6])) > 0)
				dR[8] = -(r25*pQ[7] * abs(pQ[7]) + r57*pQ[11] * abs(pQ[11]) - r47*pQ[10] * abs(pQ[10]) - r24*pQ[6] * abs(pQ[6])) / (2 * (r25*abs(pQ[7]) + r57*abs(pQ[11]) + r47*abs(pQ[10]) + r24*abs(pQ[6])));
		}
		else if (i == 9){
			if (2 * (rs0*abs(pQ[0]) + r01*abs(pQ[1]) + r16*abs(pQ[2]) + r69*abs(pQ[14]) + r9EA*abs(pQ[27])) > 0)
				dR[9] = -(rs0*pQ[0] * abs(pQ[0]) + r01*pQ[1] * abs(pQ[1]) + r16*pQ[2] * abs(pQ[2]) + r69*pQ[14] * abs(pQ[14]) + r9EA*pQ[27] * abs(pQ[27]) - 10) / (2 * (rs0*abs(pQ[0]) + r01*abs(pQ[1]) + r16*abs(pQ[2]) + r69*abs(pQ[14]) + r9EA*abs(pQ[27])));
		}
		else if (i == 10){
			if (2 * (rs0*abs(pQ[0]) + r03*abs(pQ[4]) + r3e*abs(pQ[9]) + ref*abs(pQ[18]) + rfh*abs(pQ[26]) + rhEB*abs(pQ[28])) > 0)
				dR[10] = -(rs0*pQ[0] * abs(pQ[0]) + r03*pQ[4] * abs(pQ[4]) + r3e*pQ[9] * abs(pQ[9]) + ref*pQ[18] * abs(pQ[18]) + rfh*pQ[26] * abs(pQ[26]) + rhEB*pQ[28] * abs(pQ[28]) - 10) / (2 * (rs0*abs(pQ[0]) + r03*abs(pQ[4]) + r3e*abs(pQ[9]) + ref*abs(pQ[18]) + rfh*abs(pQ[26]) + rhEB*abs(pQ[28])));
		}
		
	
		/// add the adjust flow to each pQ pipes
	

	}
}
__global__ void Add_Q(float* pQ,float*dR ,int n)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	pQ[1] = pQ[1] + dR[0] + dR[9];
	pQ[3] = pQ[3] + dR[0] - dR[1];
	pQ[6] = pQ[6] + dR[0] - dR[8];
	pQ[5] = pQ[5] - dR[0] + dR[2];
	pQ[4] = pQ[4] - dR[0] + dR[10];
	pQ[2] = pQ[2] + dR[1] + dR[9];
	pQ[8] = pQ[8] - dR[1] + dR[3];
	pQ[7] = pQ[7] - dR[1] + dR[8];
	pQ[10] = pQ[10] - dR[8] + dR[2];
	pQ[12] = pQ[12] + dR[2] - dR[3];
	pQ[15] = pQ[15] + dR[2] - dR[4];

	pQ[11] = pQ[11] - dR[3] + dR[8];
	pQ[13] = pQ[13] - dR[3] + dR[4];
	pQ[14] = pQ[14] + dR[3] + dR[9];
	pQ[23] = pQ[23] - dR[4] + dR[5];
	pQ[21] = pQ[21] - dR[4];
	pQ[17] = pQ[17] - dR[4] + dR[7];
	pQ[24] = pQ[24] + dR[5] - dR[7];
	pQ[25] = pQ[25] + dR[7];
	pQ[26] = pQ[26] - dR[7] + dR[10];
	pQ[28] = pQ[28] + dR[10];
	pQ[27] = pQ[27] + dR[9];
	pQ[19] = pQ[19] + dR[2] - dR[5];
	pQ[16] = pQ[16] - dR[2] + dR[6];
	pQ[9] = pQ[9] - dR[2] + dR[10];
	pQ[18] = pQ[18] - dR[6] + dR[10];
	pQ[20] = pQ[20] - dR[5] + dR[6];
	pQ[22] = pQ[22] + dR[6] - dR[7];
	pQ[0] = pQ[0] + dR[10] + dR[9];
}