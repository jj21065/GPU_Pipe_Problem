#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();

#endif
#include <malloc.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<windows.h>

#define N 11
#define NN 29
#define TPB 1
#define Pr 10.0
#define BPG (N+TPB-1)/TPB
float *Q; // 29
float *d_Q; // 29
float *dR;// 11
void Allocate_Memory(int n)
{
	size_t size = NN*sizeof(float);
	
	Q = (float*)malloc(size);
	hipError_t error = hipMalloc((void**)&d_Q, size);
	printf("Allocate mem dQ: %s\n", hipGetErrorString(error));
	error = hipMalloc((void**)&dR, N*sizeof(float));
	printf("Allocate mem dR : %s\n", hipGetErrorString(error));
}

void Free_Memory()
{
	if (Q)
		free(Q);
	hipError_t error = hipFree(d_Q);
	printf("Free mem dQ : %s\n", hipGetErrorString(error));
	error = hipFree(dR);
	printf("Free mem dR: %s\n", hipGetErrorString(error));
}

void CopyMemToDevice(int n)
{
	/*for (int i = 0; i < n; i++)
	{
	R[i] = data[i];
	}*/
	size_t size = NN*sizeof(float);
	hipError_t error = hipMemcpy(d_Q, Q, size, hipMemcpyHostToDevice);
	printf("Memcpy Host to Device : %s\n", hipGetErrorString(error));
}

void CopyMemToHost( int n)
{
	size_t size = NN*sizeof(float);
	hipError_t error = hipMemcpy(Q, d_Q, size, hipMemcpyDeviceToHost);
	printf("Memcpy Device to Host : %s\n", hipGetErrorString(error));
	/*for (int i = 0; i < n; i++)
	{
	data[i] = R[i];
	}*/
}

__global__ void Compute_Q(float* pQ, float*dR, int n);
__global__ void Add_Q(float* pQ, float*dR, int n);
__global__ void ErrorSum(float *error, int n);
__global__ void getError(float* dR, float *error, int n);
int main()
{
	
	
	Allocate_Memory(N);
	
	hipMemset(dR, 0.0,N*sizeof(float));

	float init_Q = sqrt(Pr);
	float Q01 = init_Q / 2.0;
	float Q24 = init_Q / 8.0;
	float Q57 = init_Q / 8.0;
	float Q16 = init_Q / 4.0;
	float Q25 = init_Q / 8.0;
	float Q78 = init_Q / 4.0;
	float Q12 = init_Q / 4.0;
	float Q56 = 0;
	float Q89 = 0;
	float Q03 = init_Q / 2.0;
	float Q3e = init_Q / 2.0;
	float Q69 = init_Q / 4.0;
	float Q34 = 0;
	float Q47 = init_Q / 8.0;
	float Q8a = init_Q / 4.0;
	float Qab = init_Q / 8.0;
	float Qad = init_Q / 8.0;
	float Qeb = 0;
	float Qbc = init_Q / 8.0;
	float Qdc = 0;
	float Qdg = init_Q / 8.0;
	float Q9g = 0;
	float Qgh = init_Q / 8.0;
	float Qef = init_Q / 2.0;
	float Qcf = init_Q / 8.0;
	float Qfh = init_Q * 5.0 / 8.0;
	float QS0 = init_Q;
	float QEA = init_Q / 4.0;
	float QEB = init_Q * 3.0 / 4.0;


	Q[0] = Q01;
	Q[1] = Q24;
	Q[2] = Q57;
	Q[3] = Q16;
	Q[4] = Q25;
	Q[5] = Q78;
	Q[6] = Q12;
	Q[7] = Q56;
	Q[8] = Q89;
	Q[9] = Q03;
	Q[10] = Q3e;
	Q[11] = Q69;
	Q[12] = Q34;
	Q[13] = Q47;
	Q[14] = Q8a;
	Q[15] = Qab;
	Q[16] = Qad;
	Q[17] = Qeb;
	Q[18] = Qbc;
	Q[19] = Qdc;
	Q[20] = Qdg;
	Q[21] = Q9g;
	Q[22] = Qgh;
	Q[23] = Qef;
	Q[24] = Qcf;
	Q[25] = Qfh;
	Q[26] = QS0;
	Q[27] = QEA;
	Q[28] = QEB;

	int n = 2;
	int iter_no = 100;
	int i;
	float h_error = 1e5;
	float*d_error;
	hipError_t tmperror = hipMalloc((void**)&d_error, BPG*sizeof(float));
	SYSTEMTIME t1,t2;
	GetLocalTime(&t1);
	
	//// main computation 

	
	CopyMemToDevice(NN);
	for (i = 0; i < 1000;i++)
	{
		Compute_Q<<<BPG,TPB>>>(d_Q,dR, N);
		getError<<<BPG,TPB>>>(dR, d_error, N);
		ErrorSum << <1, 1 >> >(d_error, BPG);
		hipMemcpy(&h_error, &(d_error[0]), sizeof(float), hipMemcpyDeviceToHost);
		if (h_error < 1e-12)
		{
			break;
		}
		Add_Q << <1, 1 >> >(d_Q, dR, N);
		
	}
	CopyMemToHost(NN);
	/////

	GetLocalTime(&t2);
	float time = t2.wSecond - t1.wSecond + (t2.wMilliseconds - t1.wMilliseconds) / 1000.0;
	printf("iter : %d, time : %g\n", i,time);
	//printf("time consume : %f\n", times);

	for (int i = 0; i < NN; i++)
	{
		printf("Q[%d] : %g\n", i ,Q[i]);
	}
	
	hipFree(d_error);
	system("pause");
	Free_Memory();
	return 0;
}
__global__ void ErrorSum(float *error,int n)
{
	int i = 0;
	for (i = 0; i < n; i++)
	{
		error[0] = error[0] + error[i];
	}
}
__global__ void getError(float* dR,float *error, int n)
{
	
	__shared__ float c[TPB];
	int I = threadIdx.x;
	int i = TPB*blockIdx.x + I;
	
	if (i < n)
		c[I] = dR[i]*dR[i];
	for (int stride = blockDim.x / 2; stride > 0; stride = stride / 2)
	{
		if (I < stride){
			c[I] += c[I + stride];
		}
		__syncthreads();
	}

	if (I == 0)
		error[blockIdx.x] = c[0];
}
__global__ void Compute_Q(float* pQ, float*dR, int n)
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	float r[29] = { 0 };
	// pipe parameter 
	float rs0 = 1;
	float r01 = 1;
	float r16 = 5;
	float r03 = 5;
	float r12 = 1;

	float r34 = 3;
	float r24 = 1;
	float r25 = 1;
	float r56 = 3;
	float r47 = 1;
	
	float r57 = 1;
	float r78 = 2;
	float r69 = 3;
	float r89 = 3;
	float r9EA = 1;

	float r3e = 1;
	float r8a = 1;
	float rab = 3;
	float rad = 2;
	float r9g = 1;

	float reb = 2;
	float rdg = 5;
	float rbc = 2;
	float rdc = 1;

	float ref = 5;
	float rcf = 2;
	float rgh = 1;
	float rfh = 2;
	float rhEB = 1;
	r[0] = r01;
	r[1] = r24;
	r[2] = r57;
	r[3] = r16;
	r[4] = r25;
	r[5] = r78;
	r[6] = r12;
	r[7] = r56;
	r[8] = r89;
	r[9] = r03;
	r[10] = r3e;
	r[11] = r69;
	r[12] = r34;
	r[13] = r47;
	r[14] = r8a;
	r[15] = rab;
	r[16] = rad;
	r[17] = reb;
	r[18] = rbc;
	r[19] = rdc;
	r[20] = rdg;
	r[21] = r9g;
	r[22] = rgh;
	r[23] = ref;
	r[24] = rcf;
	r[25] = rfh;
	r[26] = rs0;
	r[27] = r9EA;
	r[28] = rhEB;
	//float R[11] = { 0 };

	if (i < n)
	{
			// calculate adjust flow pQ
		if (i == 0)
		{
			dR[0] = -(r[0] * pQ[0] * abs(pQ[0]) + r[6] * pQ[6] * abs(pQ[6]) + r[1] * pQ[1] * abs(pQ[1]) - r[12] * pQ[12] * abs(pQ[12]) - r[9] * pQ[9] * abs(pQ[9]))
				/ (2 * r[0] * fabs(pQ[0]) + 2 * r[6] * fabs(pQ[6]) + 2 * r[1] * fabs(pQ[1]) + 2 * r[12] * fabs(pQ[12]) + 2 * r[9] * fabs(pQ[9]));
		} 
		else if (i == 1)
		{
			dR[1] = -(r[3] * pQ[3] * abs(pQ[3]) - r[7] * pQ[7] * abs(pQ[7]) - r[4] * pQ[4] * abs(pQ[4]) - r[6] * pQ[6] * abs(pQ[6]))
				/ (2 * r[3] * fabs(pQ[3]) + 2 * r[7] * fabs(pQ[7]) + 2 * r[4] * fabs(pQ[4]) + 2 * r[6] * fabs(pQ[6]));
		}
		else if (i == 2){
			dR[2] = -(r[4] * pQ[4] * abs(pQ[4]) + r[2] * pQ[2] * abs(pQ[2]) - r[13] * pQ[13] * abs(pQ[13]) - r[1] * pQ[1] * abs(pQ[1]))
				/ (2 * r[4] * fabs(pQ[4]) + 2 * r[2] * fabs(pQ[2]) + 2 * r[13] * fabs(pQ[13]) + 2 * r[1] * fabs(pQ[1]));
		}
		else if (i == 3){
			dR[3] = -(r[12] * pQ[12] * abs(pQ[12]) + r[13] * pQ[13] * abs(pQ[13]) + r[5] * pQ[5] * abs(pQ[5]) + r[14] * pQ[14] * abs(pQ[14]) + r[15] * pQ[15] * abs(pQ[15]) - r[17] * pQ[17] * abs(pQ[17]) - r[10] * pQ[10] * abs(pQ[10]))
				/ (2 * r[12] * fabs(pQ[12]) + 2 * r[13] * fabs(pQ[13]) + 2 * r[5] * fabs(pQ[5]) + 2 * r[14] * fabs(pQ[14]) + 2 * r[15] * fabs(pQ[15]) + 2 * r[17] * fabs(pQ[17]) + 2 * r[10] * fabs(pQ[10]));
		}
		else if (i == 4){
			dR[4] = -(r[7] * pQ[7] * abs(pQ[7]) + r[11] * pQ[11] * abs(pQ[11]) - r[8] * pQ[8] * abs(pQ[8]) - r[5] * pQ[5] * abs(pQ[5]) - r[2] * pQ[2] * abs(pQ[2]))
				/ (2 * r[7] * fabs(pQ[7]) + 2 * r[11] * fabs(pQ[11]) + 2 * r[8] * fabs(pQ[8]) + 2 * r[5] * fabs(pQ[5]) + 2 * r[2] * fabs(pQ[2]));
		}
		else if (i == 5){
			dR[5] = -(r[8] * pQ[8] * abs(pQ[8]) - r[21] * pQ[21] * abs(pQ[21]) - r[20] * pQ[20] * abs(pQ[20]) - r[16] * pQ[16] * abs(pQ[16]) - r[14] * pQ[14] * abs(pQ[14]))
				/ (2 * r[8] * fabs(pQ[8]) + 2 * r[21] * fabs(pQ[21]) + 2 * r[20] * fabs(pQ[20]) + 2 * r[16] * fabs(pQ[16]) + 2 * r[14] * fabs(pQ[14]));
		}
		else if (i == 6){
			dR[6] = -(r[16] * pQ[16] * abs(pQ[16]) + r[19] * pQ[19] * abs(pQ[19]) - r[18] * pQ[18] * abs(pQ[18]) - r[15] * pQ[15] * abs(pQ[15]))
				/ (2 * r[16] * fabs(pQ[16]) + 2 * r[19] * fabs(pQ[19]) + 2 * r[18] * fabs(pQ[18]) + 2 * r[15] * fabs(pQ[15]));
		}
		else if (i == 7){
			dR[7] = -(r[17] * pQ[17] * abs(pQ[17]) + r[18] * pQ[18] * abs(pQ[18]) + r[24] * pQ[24] * abs(pQ[24]) - r[23] * pQ[23] * abs(pQ[23]))
				/ (2 * r[17] * fabs(pQ[17]) + 2 * r[18] * fabs(pQ[18]) + 2 * r[24] * fabs(pQ[24]) + 2 * r[23] * fabs(pQ[23]));
		}
		else if (i == 8)
		{
			dR[8] = -(r[20] * pQ[20] * abs(pQ[20]) + r[22] * pQ[22] * abs(pQ[22]) - r[25] * pQ[25] * abs(pQ[25]) - r[24] * pQ[24] * abs(pQ[24]) - r[19] * pQ[19] * abs(pQ[19]))
				/ (2 * r[20] * fabs(pQ[20]) + 2 * r[22] * fabs(pQ[22]) + 2 * r[25] * fabs(pQ[25]) + 2 * r[24] * fabs(pQ[24]) + 2 * r[19] * fabs(pQ[19]));
		}
		else if (i == 9){
			dR[9] = -(-Pr + r[26] * pQ[26] * abs(pQ[26]) + r[0] * pQ[0] * abs(pQ[0]) + r[3] * pQ[3] * abs(pQ[3]) + r[11] * pQ[11] * abs(pQ[11]) + r[27] * pQ[27] * abs(pQ[27]))
				/ (2 * r[26] * fabs(pQ[26]) + 2 * r[0] * fabs(pQ[0]) + 2 * r[3] * fabs(pQ[3]) + 2 * r[11] * fabs(pQ[11]) + 2 * r[27] * fabs(pQ[27]));
		}
		else if (i == 10){
			dR[10] = -(-Pr + r[26] * pQ[26] * abs(pQ[26]) + r[9] * pQ[9] * abs(pQ[9]) + r[10] * pQ[10] * abs(pQ[10]) + r[23] * pQ[23] * abs(pQ[23]) + r[25] * pQ[25] * abs(pQ[25]) + r[28] * pQ[28] * abs(pQ[28]))
				/ (2 * r[26] * fabs(pQ[26]) + 2 * r[9] * fabs(pQ[9]) + 2 * r[10] * fabs(pQ[10]) + 2 * r[23] * fabs(pQ[23]) + 2 * r[25] * fabs(pQ[25]) + 2 * r[28] * fabs(pQ[28]));
		}
		
	
		/// add the adjust flow to each pQ pipes
	

	}
}
__global__ void Add_Q(float* pQ,float*dR ,int n)
{
	pQ[0] = pQ[0] + dR[0] + dR[9];
	pQ[1] = pQ[1] + dR[0] - dR[2];
	pQ[2] = pQ[2] + dR[2] - dR[4];
	pQ[3] = pQ[3] + dR[1] + dR[9];
	pQ[4] = pQ[4] - dR[1] + dR[2];
	pQ[5] = pQ[5] + dR[3] - dR[4];
	pQ[6] = pQ[6] + dR[0] - dR[1];
	pQ[7] = pQ[7] - dR[1] + dR[4];
	pQ[8] = pQ[8] - dR[4] + dR[5];
	pQ[9] = pQ[9] - dR[0] + dR[10];
	pQ[10] = pQ[10] - dR[3] + dR[10];
	pQ[11] = pQ[11] + dR[4] + dR[9];
	pQ[12] = pQ[12] - dR[0] + dR[3];
	pQ[13] = pQ[13] - dR[2] + dR[3];
	pQ[14] = pQ[14] + dR[3] - dR[5];
	pQ[15] = pQ[15] + dR[3] - dR[6];
	pQ[16] = pQ[16] + dR[6] - dR[5];
	pQ[17] = pQ[17] - dR[3] + dR[7];
	pQ[18] = pQ[18] - dR[6] + dR[7];
	pQ[19] = pQ[19] + dR[6] - dR[8];
	pQ[20] = pQ[20] - dR[5] + dR[8];
	pQ[21] = pQ[21] - dR[5];
	pQ[22] = pQ[22] + dR[8];
	pQ[23] = pQ[23] - dR[7] + dR[10];
	pQ[24] = pQ[24] + dR[7] - dR[8];
	pQ[25] = pQ[25] - dR[8] + dR[10];
	pQ[26] = pQ[26] + dR[9] + dR[10];
	pQ[27] = pQ[27] + dR[9];
	pQ[28] = pQ[28] + dR[10];


	
}