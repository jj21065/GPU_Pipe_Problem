
#include "hip/hip_runtime.h"

#include <malloc.h>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#define N 29
#define TPB 29
#define BPG (N+TPB-1)/TPB
float *Q;
float *d_Q;
void Allocate_Memory(int n)
{
	size_t size = n*sizeof(float);

	Q = (float*)malloc(size);
	hipError_t error = hipMalloc((void**)&d_Q, size);
	printf("Allocate mem : %s\n", hipGetErrorString(error));
}

void Free_Memory()
{
	if (Q)
		free(Q);
	hipError_t error = hipFree(d_Q);
	printf("Free mem : %s\n", hipGetErrorString(error));
}

void CopyMemToDevice(int n)
{
	/*for (int i = 0; i < n; i++)
	{
	R[i] = data[i];
	}*/
	size_t size = n*sizeof(float);
	hipError_t error = hipMemcpy(d_Q, Q, size, hipMemcpyHostToDevice);
	printf("Memcpy Host to Device : %s\n", hipGetErrorString(error));
}

void CopyMemToHost( int n)
{
	hipError_t error = hipMemcpy(Q, d_Q, n*sizeof(float), hipMemcpyDeviceToHost);
	printf("Memcpy Device to Host : %s\n", hipGetErrorString(error));
	/*for (int i = 0; i < n; i++)
	{
	data[i] = R[i];
	}*/
}

__global__ void Compute_Q(float* pQ, int n);

int main()
{
	
	float rs0 = 1;
	Allocate_Memory(N);

	Q[0] = sqrt(10.0 / rs0);
	Q[1] = Q[0];
	Q[2] = Q[0];
	Q[3] = 0;
	Q[4] = 0;
	Q[5] = 0;
	Q[6] = 0;
	Q[7] = 0;
	Q[8] = 0;
	Q[9] = 0;
	Q[10] = 0;
	Q[11] = 0;
	Q[12] = 0;
	Q[13] = 0;
	Q[14] = Q[0];
	Q[15] = 0;
	Q[16] = 0;
	Q[17] = 0;
	Q[18] = 0;
	Q[19] = 0;
	Q[20] = 0;
	Q[21] = 0;
	Q[22] = 0;
	Q[23] = 0;
	Q[24] = 0;
	Q[25] = 0;
	Q[26] = 0;
	Q[27] = Q[0];
	Q[28] = 0;

	int n = 2;
	int iter_no = 1;
	int i;

	printf("inital Q[0] = %g\n", Q[0]);


	clock_t t1 = clock();
	CopyMemToDevice(N);
	for (i = 0; i < iter_no; i++)
	{
		Compute_Q<<<BPG,TPB>>>(Q, N);
	}
	CopyMemToHost(N);
	clock_t t2 = clock();



	printf("time consume : %f", t2 - t1);
	//	printf("abs = %g\n",myabs(-2));
	printf("final Q[0] = %g\n", Q[0]);
	printf("final Q9-EA = %g\n", Q[27]);
	printf("final Qh-EB = %g\n", Q[28]);

	system("pause");
	Free_Memory();
	return 0;
}

__global__ void Compute_Q(float* pQ,int n )
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	// pipe parameter 
	float r01 = 1;
	float r16 = 5;
	float r12 = 1;
	float r03 = 5;
	float r34 = 3;

	float r24 = 1;
	float r25 = 1;
	float r56 = 3;
	float r3e = 1;
	float r47 = 1;

	float r57 = 1;
	float r78 = 2;
	float r89 = 3;
	float r69 = 3;

	float r8a = 1;
	float reb = 2;
	float rdg = 5;
	float ref = 5;

	float rab = 3;
	float rbc = 2;
	float r9g = 1;
	float rcf = 2;

	float rad = 2;
	float rdc = 1;
	float rgh = 1;
	float rfh = 2;

	float rs0 = 1;
	float r9EA = 1;
	float rhEB = 1;

	float R[11] = { 0 };

	if (i < n)
	{
		/// calculate adjust flow pQ
		switch (i)
		{
		case 0:
			if (2 * (r01*abs(pQ[1]) + r12*abs(pQ[3]) + r24*abs(pQ[6]) + r34*abs(pQ[5]) + r03*abs(pQ[4])) > 0)
				R[0] = -(r01*pQ[1] * abs(pQ[1]) + r12*pQ[3] * abs(pQ[3]) + r24*pQ[6] * abs(pQ[6]) - r34*pQ[5] * abs(pQ[5]) - r03*pQ[4] * abs(pQ[4])) / (2 * (r01*abs(pQ[1]) + r12*abs(pQ[3]) + r24*abs(pQ[6]) + r34*abs(pQ[5]) + r03*abs(pQ[4])));
			break;
		case 1:
			if (2 * (r16*abs(pQ[2]) + r56*abs(pQ[8]) + r25*abs(pQ[7]) + r12*abs(pQ[3])) > 0)
				R[1] = -(r16*pQ[2] * abs(pQ[2]) - r56*pQ[8] * abs(pQ[8]) - r25*pQ[7] * abs(pQ[7]) - r12*pQ[3] * abs(pQ[3])) / (2 * (r16*abs(pQ[2]) + r56*abs(pQ[8]) + r25*abs(pQ[7]) + r12*abs(pQ[3])));
			break;
		case 2:
			if (2 * (r34*abs(pQ[5]) + r47*abs(pQ[10]) + r78*abs(pQ[12]) + r8a*abs(pQ[15]) + rab*abs(pQ[19]) + reb*abs(pQ[16]) + r3e*abs(pQ[9])) > 0)
				R[2] = -(r34*pQ[5] * abs(pQ[5]) + r47*pQ[10] * abs(pQ[10]) + r78*pQ[12] * abs(pQ[12]) + r8a*pQ[15] * abs(pQ[15]) + rab*pQ[19] * abs(pQ[19]) - reb*pQ[16] * abs(pQ[16]) - r3e*pQ[9] * abs(pQ[9])) / (2 * (r34*abs(pQ[5]) + r47*abs(pQ[10]) + r78*abs(pQ[12]) + r8a*abs(pQ[15]) + rab*abs(pQ[19]) + reb*abs(pQ[16]) + r3e*abs(pQ[9])));
			break;
		case 3:
			if (2 * (r56*abs(pQ[8]) + r69*abs(pQ[14]) + r89*abs(pQ[13]) + r78*abs(pQ[12]) + r57*abs(pQ[11])) > 0)
				R[3] = -(r56*pQ[8] * abs(pQ[8]) + r69*pQ[14] * abs(pQ[14]) - r89*pQ[13] * abs(pQ[13]) - r78*pQ[12] * abs(pQ[12]) - r57*pQ[11] * abs(pQ[11])) / (2 * (r56*abs(pQ[8]) + r69*abs(pQ[14]) + r89*abs(pQ[13]) + r78*abs(pQ[12]) + r57*abs(pQ[11])));
			break;
		case 4:
			if (2 * (r89*abs(pQ[13]) + r9g*abs(pQ[21]) + rdg*abs(pQ[17]) + rad*abs(pQ[23]) + r8a*abs(pQ[15])) > 0)
				R[4] = -(r89*pQ[13] * abs(pQ[13]) - r9g*pQ[21] * abs(pQ[21]) - rdg*pQ[17] * abs(pQ[17]) - rad*pQ[23] * abs(pQ[23]) - r8a*pQ[15] * abs(pQ[15])) / (2 * (r89*abs(pQ[13]) + r9g*abs(pQ[21]) + rdg*abs(pQ[17]) + rad*abs(pQ[23]) + r8a*abs(pQ[15])));
			break;
		case 5:
			if (2 * (rad*abs(pQ[23]) + rdc*abs(pQ[24]) + rbc*abs(pQ[20]) + rab*abs(pQ[19])) > 0)
				R[5] = -(rad*pQ[23] * abs(pQ[23]) + rdc*pQ[24] * abs(pQ[24]) - rbc*pQ[20] * abs(pQ[20]) - rab*pQ[19] * abs(pQ[19])) / (2 * (rad*abs(pQ[23]) + rdc*abs(pQ[24]) + rbc*abs(pQ[20]) + rab*abs(pQ[19])));
			break;
		case 6:

			if (2 * (reb*abs(pQ[16]) + rbc*abs(pQ[20]) + rcf*abs(pQ[22]) + ref*abs(pQ[18])) > 0)
				R[6] = -(reb*pQ[16] * abs(pQ[16]) + rbc*pQ[20] * abs(pQ[20]) + rcf*pQ[22] * abs(pQ[22]) - ref*pQ[18] * abs(pQ[18])) / (2 * (reb*abs(pQ[16]) + rbc*abs(pQ[20]) + rcf*abs(pQ[22]) + ref*abs(pQ[18])));
			break;
		case 7:
			if (2 * (rdg*abs(pQ[17]) + rgh*abs(pQ[25]) + rfh*abs(pQ[26]) + rcf*abs(pQ[22]) + rdc*abs(pQ[24])) > 0)
				R[7] = -(rdg*pQ[17] * abs(pQ[17]) + rgh*pQ[25] * abs(pQ[25]) - rfh*pQ[26] * abs(pQ[26]) - rcf*pQ[22] * abs(pQ[22]) - rdc*pQ[24] * abs(pQ[24])) / (2 * (rdg*abs(pQ[17]) + rgh*abs(pQ[25]) + rfh*abs(pQ[26]) + rcf*abs(pQ[22]) + rdc*abs(pQ[24])));
			break;
		case 8:
			if (2 * (r25*abs(pQ[7]) + r57*abs(pQ[11]) + r47*abs(pQ[10]) + r24*abs(pQ[6])) > 0)
				R[8] = -(r25*pQ[7] * abs(pQ[7]) + r57*pQ[11] * abs(pQ[11]) - r47*pQ[10] * abs(pQ[10]) - r24*pQ[6] * abs(pQ[6])) / (2 * (r25*abs(pQ[7]) + r57*abs(pQ[11]) + r47*abs(pQ[10]) + r24*abs(pQ[6])));
			break;
		case 9:
			if (2 * (rs0*abs(pQ[0]) + r01*abs(pQ[1]) + r16*abs(pQ[2]) + r69*abs(pQ[14]) + r9EA*abs(pQ[27])) > 0)
				R[9] = -(rs0*pQ[0] * abs(pQ[0]) + r01*pQ[1] * abs(pQ[1]) + r16*pQ[2] * abs(pQ[2]) + r69*pQ[14] * abs(pQ[14]) + r9EA*pQ[27] * abs(pQ[27]) - 10) / (2 * (rs0*abs(pQ[0]) + r01*abs(pQ[1]) + r16*abs(pQ[2]) + r69*abs(pQ[14]) + r9EA*abs(pQ[27])));
			break;
		case 10:
			if (2 * (rs0*abs(pQ[0]) + r03*abs(pQ[4]) + r3e*abs(pQ[9]) + ref*abs(pQ[18]) + rfh*abs(pQ[26]) + rhEB*abs(pQ[28])) > 0)
				R[10] = -(rs0*pQ[0] * abs(pQ[0]) + r03*pQ[4] * abs(pQ[4]) + r3e*pQ[9] * abs(pQ[9]) + ref*pQ[18] * abs(pQ[18]) + rfh*pQ[26] * abs(pQ[26]) + rhEB*pQ[28] * abs(pQ[28]) - 10) / (2 * (rs0*abs(pQ[0]) + r03*abs(pQ[4]) + r3e*abs(pQ[9]) + ref*abs(pQ[18]) + rfh*abs(pQ[26]) + rhEB*abs(pQ[28])));
			break;

		}
	
		/// add the adjust flow to each pQ pipes
		pQ[1] = pQ[1] + R[0] + R[9];
		pQ[3] = pQ[3] + R[0] - R[1];
		pQ[6] = pQ[6] + R[0] - R[8];
		pQ[5] = pQ[5] - R[0] + R[2];
		pQ[4] = pQ[4] - R[0] + R[10];
		pQ[2] = pQ[2] + R[1] + R[9];
		pQ[8] = pQ[8] - R[1] + R[3];
		pQ[7] = pQ[7] - R[1] + R[8];
		pQ[10] = pQ[10] - R[8] + R[2];
		pQ[12] = pQ[12] + R[2] - R[3];
		pQ[15] = pQ[15] + R[2] - R[4];

		pQ[11] = pQ[11] - R[3] + R[8];
		pQ[13] = pQ[13] - R[3] + R[4];
		pQ[14] = pQ[14] + R[3] + R[9];
		pQ[23] = pQ[23] - R[4] + R[5];
		pQ[21] = pQ[21] - R[4];
		pQ[17] = pQ[17] - R[4] + R[7];
		pQ[24] = pQ[24] + R[5] - R[7];
		pQ[25] = pQ[25] + R[7];
		pQ[26] = pQ[26] - R[7] + R[10];
		pQ[28] = pQ[28] + R[10];
		pQ[27] = pQ[27] + R[9];
		pQ[19] = pQ[19] + R[2] - R[5];
		pQ[16] = pQ[16] - R[2] + R[6];
		pQ[9] = pQ[9] - R[2] + R[10];
		pQ[18] = pQ[18] - R[6] + R[10];
		pQ[20] = pQ[20] - R[5] + R[6];
		pQ[22] = pQ[22] + R[6] - R[7];
		pQ[0] = pQ[0] + R[10] + R[9];

	}
}