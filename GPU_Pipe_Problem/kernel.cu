#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include<windows.h>

#define N 29
float rs0 = 1;
float r01 = 1;
float r16 = 5;
float r03 = 5;
float r12 = 1;

float r34 = 3;
float r24 = 1;
float r25 = 1;
float r56 = 3;
float r47 = 1;
float r57 = 1;
float r78 = 2;
float r69 = 3;
float r89 = 3;
float r9EA = 1;
float r3e = 1;
float r8a = 1;
float rab = 3;
float rad = 2;
float r9g = 1;

float reb = 2;
float rdg = 5;
float rbc = 2;
float rdc = 1;

float ref = 5;
float rcf = 2;
float rgh = 1;
float rfh = 2;
float rhEB = 1;


void MatrixVectorProduct(float *a, float*p, float *ap, int n);

float Dot(float *v1, float *v2, int n);

void Sum_Store(float *store, float *x, float scalar, float*v, int n);

void conjgrad(float*A, float*x, float*b);

void Newton_Raphson_method(float *x, float*r);

void Compute_R(float*x, float*r, float *R);

void Compute_J(float*x, float*r, int n);

void Compute_invJ_mul_R(float*x, float*r, float*R, float*invJ_R);

void inverse(float*A, float* invA, int n);

float Hardy_Cross_method(float* Q);

int main()
{
	float* Q = new float[N];
	float* r = new float[N];
	// pipe parameter 
	Q[0] = sqrt(10.0 / rs0);
	Q[1] = Q[0];
	Q[2] = Q[0];
	Q[3] = 0;
	Q[4] = 0;
	Q[5] = 0;
	Q[6] = 0;
	Q[7] = 0;
	Q[8] = 0;
	Q[9] = 0;
	Q[10] = 0;

	Q[11] = 0;
	Q[12] = 0;
	Q[13] = 0;
	Q[14] = Q[0];

	Q[15] = 0;
	Q[16] = 0;
	Q[17] = 0;
	Q[18] = 0;

	Q[19] = 0;
	Q[20] = 0;
	Q[21] = 0;
	Q[22] = 0;

	Q[23] = 0;
	Q[24] = 0;
	Q[25] = 0;
	Q[26] = 0;
	
	Q[27] = Q[0];
	Q[28] = 0;

	/*Q[0] = 1;
	Q[1] = 1;
	Q[2] = 1;
	Q[3] = 1;
	Q[4] = 1;
	Q[5] = 1;
	Q[6] = 1;
	Q[7] = 1;
	Q[8] = 1;
	Q[9] = 1;
	Q[10] = 1;

	Q[11] = 1;
	Q[12] = 1;
	Q[13] = 1;
	Q[14] = 1;

	Q[15] = 1;
	Q[16] =1;
	Q[17] = 1;
	Q[18] =1;

	Q[19] = 1;
	Q[20] = 1;
	Q[21] = 1;
	Q[22] = 1;

	Q[23] = 1;
	Q[24] = 1;
	Q[25] = 1;
	Q[26] = 1;

	Q[27] = 1;
	Q[28] = 1;*/

	r[0] = 1;
	r[1] = 1;
	r[2] = 5;
	r[3] = 5;
	r[4] = 1;

	r[5] = 3;
	r[6] = 1;
	r[7] = 1;
	r[8] = 3;
	r[9] = 1;
	
	r[10] = 1;
	r[11] = 2;
	r[12] = 3;
	r[13] = 3;
	r[14] = 1;

	r[15] = 1;
	r[16] = 1;
	r[17] = 3;
	r[18] = 2;
	r[19] = 1;

	r[20] = 2;
	r[21] = 5;
	r[22] = 2;
	r[23] = 1;

	r[24] = 5;
	r[25] = 2;
	r[26] = 1;
	r[27] = 2;
	r[28] = 1;

	// initial conditions 

	int iter_no = 1000;
	int i;
	float Error = 1e5;

	SYSTEMTIME t1, t2;
	GetLocalTime(&t1);

	for (i = 0; i < iter_no; i++)
	{
		Error = Hardy_Cross_method(Q);
		/*if (Error < 1e-8)
			break;*/
		//Newton_Raphson_method(Q, r);
	}
	GetLocalTime(&t2);
	float time = t2.wSecond - t1.wSecond + (t2.wMilliseconds - t1.wMilliseconds) / 1000.0;
	printf("iter : %d, time : %g\n", i, time);
	for (int i = 0; i < 29; i++)
		printf("final Q[%d] = %g\n", i, Q[i]);

	/*
		printf("final Q[0] = %g\n", Q[0]);
		printf("final Q9-EA = %g\n", Q[27]);
		printf("final Qh-EB = %g\n", Q[28]);*/
	system("pause");

	delete[] Q;
	delete[] r;

	return 0;
}

float Hardy_Cross_method(float* Q)
{
	float R[11] = { 0 };

	/// calculate adjust flow Q
	if (2 * (r01*abs(Q[1]) + r12*abs(Q[3]) + r24*abs(Q[6]) + r34*abs(Q[5]) + r03*abs(Q[4])) > 0)
		R[0] = -(r01*Q[1] * abs(Q[1]) + r12*Q[3] * abs(Q[3]) + r24*Q[6] * abs(Q[6]) - r34*Q[5] * abs(Q[5]) - r03*Q[4] * abs(Q[4])) / (2 * (r01*abs(Q[1]) + r12*abs(Q[3]) + r24*abs(Q[6]) + r34*abs(Q[5]) + r03*abs(Q[4])));

	if (2 * (r16*abs(Q[2]) + r56*abs(Q[8]) + r25*abs(Q[7]) + r12*abs(Q[3])) > 0)
		R[1] = -(r16*Q[2] * abs(Q[2]) - r56*Q[8] * abs(Q[8]) - r25*Q[7] * abs(Q[7]) - r12*Q[3] * abs(Q[3])) / (2 * (r16*abs(Q[2]) + r56*abs(Q[8]) + r25*abs(Q[7]) + r12*abs(Q[3])));

	if (2 * (r34*abs(Q[5]) + r47*abs(Q[10]) + r78*abs(Q[12]) + r8a*abs(Q[15]) + rab*abs(Q[19]) + reb*abs(Q[16]) + r3e*abs(Q[9])) > 0)
		R[2] = -(r34*Q[5] * abs(Q[5]) + r47*Q[10] * abs(Q[10]) + r78*Q[12] * abs(Q[12]) + r8a*Q[15] * abs(Q[15]) + rab*Q[19] * abs(Q[19]) - reb*Q[16] * abs(Q[16]) - r3e*Q[9] * abs(Q[9])) / (2 * (r34*abs(Q[5]) + r47*abs(Q[10]) + r78*abs(Q[12]) + r8a*abs(Q[15]) + rab*abs(Q[19]) + reb*abs(Q[16]) + r3e*abs(Q[9])));

	if (2 * (r56*abs(Q[8]) + r69*abs(Q[14]) + r89*abs(Q[13]) + r78*abs(Q[12]) + r57*abs(Q[11])) > 0)
		R[3] = -(r56*Q[8] * abs(Q[8]) + r69*Q[14] * abs(Q[14]) - r89*Q[13] * abs(Q[13]) - r78*Q[12] * abs(Q[12]) - r57*Q[11] * abs(Q[11])) / (2 * (r56*abs(Q[8]) + r69*abs(Q[14]) + r89*abs(Q[13]) + r78*abs(Q[12]) + r57*abs(Q[11])));

	if (2 * (r89*abs(Q[13]) + r9g*abs(Q[21]) + rdg*abs(Q[17]) + rad*abs(Q[23]) + r8a*abs(Q[15])) > 0)
		R[4] = -(r89*Q[13] * abs(Q[13]) - r9g*Q[21] * abs(Q[21]) - rdg*Q[17] * abs(Q[17]) - rad*Q[23] * abs(Q[23]) - r8a*Q[15] * abs(Q[15])) / (2 * (r89*abs(Q[13]) + r9g*abs(Q[21]) + rdg*abs(Q[17]) + rad*abs(Q[23]) + r8a*abs(Q[15])));

	if (2 * (rad*abs(Q[23]) + rdc*abs(Q[24]) + rbc*abs(Q[20]) + rab*abs(Q[19])) > 0)
		R[5] = -(rad*Q[23] * abs(Q[23]) + rdc*Q[24] * abs(Q[24]) - rbc*Q[20] * abs(Q[20]) - rab*Q[19] * abs(Q[19])) / (2 * (rad*abs(Q[23]) + rdc*abs(Q[24]) + rbc*abs(Q[20]) + rab*abs(Q[19])));

	if (2 * (reb*abs(Q[16]) + rbc*abs(Q[20]) + rcf*abs(Q[22]) + ref*abs(Q[18])) > 0)
		R[6] = -(reb*Q[16] * abs(Q[16]) + rbc*Q[20] * abs(Q[20]) + rcf*Q[22] * abs(Q[22]) - ref*Q[18] * abs(Q[18])) / (2 * (reb*abs(Q[16]) + rbc*abs(Q[20]) + rcf*abs(Q[22]) + ref*abs(Q[18])));

	if (2 * (rdg*abs(Q[17]) + rgh*abs(Q[25]) + rfh*abs(Q[26]) + rcf*abs(Q[22]) + rdc*abs(Q[24])) > 0)
		R[7] = -(rdg*Q[17] * abs(Q[17]) + rgh*Q[25] * abs(Q[25]) - rfh*Q[26] * abs(Q[26]) - rcf*Q[22] * abs(Q[22]) - rdc*Q[24] * abs(Q[24])) / (2 * (rdg*abs(Q[17]) + rgh*abs(Q[25]) + rfh*abs(Q[26]) + rcf*abs(Q[22]) + rdc*abs(Q[24])));

	if (2 * (r25*abs(Q[7]) + r57*abs(Q[11]) + r47*abs(Q[10]) + r24*abs(Q[6])) > 0)
		R[8] = -(r25*Q[7] * abs(Q[7]) + r57*Q[11] * abs(Q[11]) - r47*Q[10] * abs(Q[10]) - r24*Q[6] * abs(Q[6])) / (2 * (r25*abs(Q[7]) + r57*abs(Q[11]) + r47*abs(Q[10]) + r24*abs(Q[6])));

	if (2 * (rs0*abs(Q[0]) + r01*abs(Q[1]) + r16*abs(Q[2]) + r69*abs(Q[14]) + r9EA*abs(Q[27])) > 0)
		R[9] = -(rs0*Q[0] * abs(Q[0]) + r01*Q[1] * abs(Q[1]) + r16*Q[2] * abs(Q[2]) + r69*Q[14] * abs(Q[14]) + r9EA*Q[27] * abs(Q[27]) - 10) / (2 * (rs0*abs(Q[0]) + r01*abs(Q[1]) + r16*abs(Q[2]) + r69*abs(Q[14]) + r9EA*abs(Q[27])));

	if (2 * (rs0*abs(Q[0]) + r03*abs(Q[4]) + r3e*abs(Q[9]) + ref*abs(Q[18]) + rfh*abs(Q[26]) + rhEB*abs(Q[28])) > 0)
		R[10] = -(rs0*Q[0] * abs(Q[0]) + r03*Q[4] * abs(Q[4]) + r3e*Q[9] * abs(Q[9]) + ref*Q[18] * abs(Q[18]) + rfh*Q[26] * abs(Q[26]) + rhEB*Q[28] * abs(Q[28]) - 10) / (2 * (rs0*abs(Q[0]) + r03*abs(Q[4]) + r3e*abs(Q[9]) + ref*abs(Q[18]) + rfh*abs(Q[26]) + rhEB*abs(Q[28])));

	/// add the adjust flow to each Q pipes
	Q[1] = Q[1] + R[0] + R[9];
	Q[3] = Q[3] + R[0] - R[1];
	Q[6] = Q[6] + R[0] - R[8];
	Q[5] = Q[5] - R[0] + R[2];
	Q[4] = Q[4] - R[0] + R[10];
	Q[2] = Q[2] + R[1] + R[9];
	Q[8] = Q[8] - R[1] + R[3];
	Q[7] = Q[7] - R[1] + R[8];
	Q[10] = Q[10] - R[8] + R[2];
	Q[12] = Q[12] + R[2] - R[3];
	Q[15] = Q[15] + R[2] - R[4];

	Q[11] = Q[11] - R[3] + R[8];
	Q[13] = Q[13] - R[3] + R[4];
	Q[14] = Q[14] + R[3] + R[9];
	Q[23] = Q[23] - R[4] + R[5];
	Q[21] = Q[21] - R[4];
	Q[17] = Q[17] - R[4] + R[7];
	Q[24] = Q[24] + R[5] - R[7];
	Q[25] = Q[25] + R[7];
	Q[26] = Q[26] - R[7] + R[10];
	Q[28] = Q[28] + R[10];
	Q[27] = Q[27] + R[9];
	Q[19] = Q[19] + R[2] - R[5];
	Q[16] = Q[16] - R[2] + R[6];
	Q[9] = Q[9] - R[2] + R[10];
	Q[18] = Q[18] - R[6] + R[10];
	Q[20] = Q[20] - R[5] + R[6];
	Q[22] = Q[22] + R[6] - R[7];
	Q[0] = Q[0] + R[10] + R[9];

	float tmpErr = 0;
	for (int i = 0; i < 11; i++)
	{
		tmpErr = tmpErr + R[i] * R[i];
	}
	return tmpErr;

}

void Newton_Raphson_method(float *x, float*r)
{

	/// Computet R 
	float invJR[N] = { 0 };
	float R[N] = { 0 };
	Compute_R(x, r, R);
	/// Compute J 
	Compute_invJ_mul_R(x, r, R, invJR);

	for (int i = 0; i < N; i++)
	{
		x[i] = x[i] - invJR[i];
	}
	// x = x - inv(J')*R'
}

void Compute_R(float*x, float*r, float *R)
{

	R[0] = (r01*x[1] * abs(x[1]) + r12*x[4] * abs(x[4]) + r24*x[6] * abs(x[6]) - r34*x[5] * abs(x[5]) - r03*x[3] * abs(x[3]));

	R[1] = (r16*x[2] * abs(x[2]) - r56*x[8] * abs(x[8]) - r25*x[7] * abs(x[7]) - r12*x[4] * abs(x[4]));

	R[2] = (r25*x[7] * abs(x[7]) + r57*x[10] * abs(x[10]) - r47*x[9] * abs(x[9]) - r24*x[6] * abs(x[6]));

	R[3] = (r34*x[5] * abs(x[5]) + r47*x[9] * abs(x[9]) + r78*x[11] * abs(x[11]) + r8a*x[16] * abs(x[16]) + rab*x[17] * abs(x[17]) - reb*x[20] * abs(x[20]) - r3e*x[15] * abs(x[15]));

	R[4] = (r56*x[8] * abs(x[8]) + r69*x[12] * abs(x[12]) - r89*x[13] * abs(x[13]) - r78*x[11] * abs(x[11]) - r57*x[10] * abs(x[10]));

	R[5] = (r89*x[13] * abs(x[13]) - r9g*x[19] * abs(x[19]) - rdg*x[21] * abs(x[21]) - rad*x[18] * abs(x[18]) - r8a*x[16] * abs(x[16]));

	R[6] = (rad*x[18] * abs(x[18]) + rdc*x[23] * abs(x[23]) - rbc*x[22] * abs(x[22]) - rab*x[17] * abs(x[17]));

	R[7] = (reb*x[20] * abs(x[20]) + rbc*x[22] * abs(x[22]) + rcf*x[25] * abs(x[25]) - ref*x[24] * abs(x[24]));

	R[8] = (rdg*x[21] * abs(x[21]) + rgh*x[26] * abs(x[26]) - rfh*x[27] * abs(x[27]) - rcf*x[25] * abs(x[25]) - rdc*x[23] * abs(x[23]));

	R[9] = x[25] + x[24] - x[27];

	R[10] = (rs0*x[0] * abs(x[0]) + r01*x[1] * abs(x[1]) + r16*x[2] * abs(x[2]) + r69*x[12] * abs(x[12]) + r9EA*x[14] * abs(x[14]) - 10);

	R[11] = (rs0*x[0] * abs(x[0]) + r03*x[3] * abs(x[3]) + r3e*x[15] * abs(x[15]) + ref*x[24] * abs(x[24]) + rfh*x[27] * abs(x[27]) + rhEB*x[28] * abs(x[28]) - 10);

	R[12] = x[0] - x[1] - x[3];

	R[13] = x[1] - x[4] - x[2];

	R[14] = x[4] - x[6] - x[7];

	R[15] = x[3] - x[5] - x[15];

	R[16] = x[5] + x[6] - x[9];

	R[17] = x[7] - x[8] - x[10];

	R[18] = x[2] + x[8] - x[12];

	R[19] = x[10] + x[9] - x[11];

	R[20] = x[11] - x[13] - x[16];

	R[21] = x[13] + x[12] + x[19] - x[14];

	R[22] = x[16] - x[17] - x[18];

	R[23] = x[15] - x[20] - x[24];

	R[24] = x[20] + x[17] - x[22];

	R[25] = x[18] - x[21] - x[23];

	R[26] = x[21] + x[19] - x[26];

	R[27] = x[26] + x[27] - x[28];

	R[28] = x[22] + x[23] - x[25];


}

void Compute_invJ_mul_R(float*x, float*r, float*R, float*invJ_R)
{
	/// Conjugate gradient method 
	float A[N*N] = { 0 };

	A[0] = 0; A[1] = 2 * r[1] * x[1];  A[4] = 2 * r[4] * x[4]; A[6] = 2 * r[6] * x[6]; A[5] = -2 * r[5] * x[5]; A[3] = -2 * r[3] * x[3];

	A[29] = 0; A[29 + 2] = 2 * r[2] * x[2]; A[29 + 8] = -2 * r[8] * x[8]; A[29 + 7] = -2 * r[7] * x[7]; A[29 + 4] = -2 * r[4] * x[4];

	A[58] = 0; A[58 + 7] = 2 * r[7] * x[7]; A[58 + 10] = 2 * r[10] * x[10]; A[58 + 9] = -2 * r[9] * x[9]; A[58 + 6] = -2 * r[6] * x[6];

	A[87] = 0; A[87 + 5] = 2 * r[5] * x[5]; A[9] = 2 * r[9] * x[9]; A[87 + 11] = 2 * r[11] * x[11]; A[87 + 16] = 2 * r[16] * x[16]; A[87 + 17] = 2 * r[17] * x[17]; A[87 + 20] = -2 * r[20] * x[20]; A[87 + 15] = -2 * r[15] * x[15];

	A[116] = 0; A[116 + 8] = 2 * r[8] * x[8]; A[116 + 12] = 2 * r[12] * x[12]; A[116 + 13] = -2 * r[13] * x[13]; A[116 + 11] = -2 * r[11] * x[11]; A[116 + 10] = -2 * r[10] * x[10];
	
	///
	
	A[145] = 0; A[145 + 13] = 2 * r[13] * x[13]; A[145 + 19] = -2 * r[19] * x[19]; A[145 + 21] = -2 * r[21] * x[21]; A[145 + 18] = -2 * r[18] * x[18]; A[145 + 16] = -2 * r[16] * x[16];

	A[174] = 0; A[174 + 18] = 2 * r[18] * x[18]; A[174 + 23] = 2 * r[23] * x[23]; A[174 + 22] = -2 * r[22] * x[22]; A[174 + 17] = -2 * r[17] * x[17];

	A[203] = 0; A[203 + 20] = 2 * r[20] * x[20]; A[203 + 22] = 2 * r[22] * x[22]; A[203 + 25] = 2 * r[25] * x[25]; A[203 + 24] = -2 * r[24] * x[24];

	A[232] = 0; A[232 + 21] = 2 * r[21] * x[21]; A[232 + 26] = 2 * r[26] * x[26]; A[232 + 27] = -2 * r[27] * x[27]; A[232 + 25] = -2 * r[25] * x[25]; A[232 + 23] = -2 * r[23] * x[23];

	A[261] = 0; A[261 + 25] = 1; A[261 + 24] = 1; A[261 + 27] = -1;

	///

	A[290] = 2 * r[0] * x[0]; A[290 + 1] = 2 * r[1] * x[1]; A[290 + 2] = 2 * r[2] * x[2]; A[290 + 12] = 2 * r[12] * x[12]; A[290 + 14] = 2 * r[14] * x[14];

	A[319] = 2 * r[0] * x[0]; A[319 + 3] = 2 * r[3] * x[3]; A[319 + 15] = 2 * r[15] * x[15]; A[319 + 24] = 2 * r[24] * x[24]; A[319 + 27] = 2 * r[27] * x[27]; A[319 + 28] = 2 * r[28] * x[28];

	A[348] = 1; A[348 + 1] = -1; A[348 + 3] = -1;

	A[377] = 0; A[377 + 1] = 1; A[377 + 4] = -1; A[377 + 2] = 1;

	A[406] = 0; A[406+4] = 1; A[406+6] = -1; A[406+7] = -1;

	A[435] = 0; A[435+3] = 1; A[435+5] = -1; A[435+15] = -1;

	A[464] = 0; A[464+5] = 1; A[464+6] = 1; A[464+9] = -1;

	A[493] = 0; A[493+7] = 1; A[493+8] = 1; A[493+10] = -1;

	A[522] = 0; A[522+2] = 1; A[522+8] = 1; A[522+12] = -1;

	A[551] = 0; A[551+10] = 1; A[551+9] = 1; A[551+11] = -1;

	A[580] = 0; A[580+11] = 1; A[580+13] = -1; A[580+16] = -1;

	A[609] = 0; A[609+13] = 1; A[609+12] = 1; A[609+19] = 1; A[609+14] = -1;

	A[638] = 0; A[638+16] = 1; A[638+17] = -1; A[638+18] = -1;

	A[667] = 0; A[667+15] = 1; A[667+20] = -1; A[667+24] = -1;

	A[696] = 0; A[696+20] = 1; A[696+17] = 1; A[696+22] = -1;

	A[725] = 0; A[725+18] = 1; A[725+21] = -1; A[725+23] = -1;

	A[754] = 0; A[754+21] = 1; A[754+19] = 1; A[754+26] = -1;

	A[783] = 0; A[783+26] = 1; A[783+27] = 1; A[783+28] = -1;

	A[812] = 0; A[812+22] = 1; A[812+23] = 1; A[812+25] = -1;

	//FILE*pfile;
	//pfile = fopen("output.txt", "w");

	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//	{
	//		fprintf(pfile, "%g\t", A[i*N + j]);
	//	}
	//	fprintf(pfile, "\n");
	//}
	//fclose(pfile);
	//	conjgrad(A, invJ_R, R);
	float Atemp[N*N] = { 0 };
	//inverse(A, Atemp, N);
	conjgrad(A, invJ_R, R);
	//MatrixVectorProduct(Atemp, R, invJ_R, N);

}

void MatrixVectorProduct(float *a, float*p, float *ap, int n)
{
	int i = 0;
	int j = 0;

	for (i = 0; i < n; i++){
		ap[i] = 0.0;
		for (j = 0; j < n; j++){
			ap[i] = ap[i] + a[i*n + j] * p[j];
		}
	}
}

float Dot(float *v1, float *v2, int n)
{
	int i;
	float ans = 0.0;
	for (i = 0; i < n; i++){
		ans += v1[i] * v2[i];

	}
	return ans;
}

void Sum_Store(float *store, float *x, float scalar, float*v, int n)
{
	int i = 0;
	for (i = 0; i < n; i++){
		store[i] = x[i] + scalar * v[i];
	}
}

void conjgrad(float*A, float*x, float*b)
{
	float ap[N];
	float p[N];
	float rr[N];
	float rsold = 0;
	float rsnew = 0;
	int i = 0;
	MatrixVectorProduct(A, x, ap, N);
	for (i = 0; i < N; i++){
		rr[i] = b[i] - ap[i];
		p[i] = rr[i];
	}
	rsold = Dot(rr, rr, N);

	for (i = 0; i < N; i++){
		MatrixVectorProduct(A, p, ap, N);
		/*	for (int j = 0; j < N*N; j++){
				printf("ap[%d] = %g\n", j, ap[j]);
				}*/
		float tmpap = Dot(ap, p, N);
		float alpha = rsold / (tmpap);
		Sum_Store(x, x, alpha, p, N);
		Sum_Store(rr, rr, -alpha, ap, N);
		rsnew = Dot(rr, rr, N);
		if (sqrt(rsnew) < 1e-10)
			break;
		Sum_Store(p, rr, (rsnew / rsold), p, N);
		rsold = rsnew;

	}
	// 	for (i = 0; i < N; i++)
	// 		printf("x[%d] = %g\n", i, x[i]);


}

void inverse(float*A, float* invA, int n)
{
	int i = 0, j = 0, k = 0;
	float d;
	float *a = new float[n*n * 2];

	for (i = 0; i < n*n * 2; i++)
		a[i] = 0;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			a[i*n * 2 + j] = A[i*n + j];
		}
	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	for (i = 0; i < n; i++)
		for (j = 0; j < 2 * n; j++)
			if (j == (i + n))
				a[i*n * 2 + j] = 1;

	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	/************** partial pivoting **************/
	for (i = n - 1; i > 0; i--)
	{
		if (a[(i - 1)*n * 2 + 0] < a[i*n * 2 + 0])
			for (j = 0; j < n * 2; j++)
			{
				d = a[i*n * 2 + j];
				a[i*n * 2 + j] = a[(i - 1)*n * 2 + j];
				a[(i - 1)*n * 2 + j] = d;
			}
	}
	//cout << "pivoted output: " << endl;
	/*for (i = 0; i < n; i++)
	{
	for (j = 0; j < 2 * n; j++)
	{
	printf("%g\t", a[i*n*2 + j]);
	}
	printf("\n");
	}*/
	/********** reducing to diagonal  matrix ***********/

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
			if (j != i)
			{
				d = a[j*n * 2 + i] / a[i*n * 2 + i];
				for (k = 0; k < n * 2; k++)
					a[j*n * 2 + k] -= a[i*n * 2 + k] * d;
			}
	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n * 2 + j]);
	//	}
	//	printf("\n");
	//}
	/************** reducing to unit matrix *************/
	for (i = 0; i < n; i++)
	{
		d = a[i*n * 2 + i];
		if (d != 0)
			for (j = 0; j < n * 2; j++)
				a[i*n * 2 + j] = a[i*n * 2 + j] / d;
	}

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
			invA[i*n + j] = a[i*n * 2 + n + j];

	}
	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j < 2 * n; j++)
	//	{
	//		printf("%g\t", a[i*n*2 + j]);
	//	}
	//	printf("\n");
	//}

	//for (i = 0; i < n; i++)
	//{
	//	for (j = 0; j <  n; j++)
	//	{
	//		printf("%g\t", invA[i*n+j]);
	//	}
	//	printf("\n");
	//}
	delete[] a;
}

float norm(float *a, int n)
{

	float value = 0;
	for (int i = 0; i < n; i++)
	{
		value = value + a[i] * a[i];
	}
	return value;
}

//void BICG(float *A, float *x, float *b)
//{
//	int iter = N;
//	int flag = 0;
//
//	float norm2 = norm(b, N);
//	if (norm2 == 0.0)
//		norm2 = 1.0;
//	float ap[N] = { 0 };
//	float r[N] = { 0 };
//	MatrixVectorProduct(A, x, ap, N);
//	for (int i = 0; i < N; i++){
//		r[i] = b[i] - ap[i];
//	}
//		
//	float error = norm(r, N) / norm2;
//	if (error < 0.001) 
//		return;
//
//	float r_tld[N];
//	for (int i = 0; i < N; i++)
//		r_tld[i] = r[i];
//
//	for(int i = 0;i<iter;i++)
//		                   
//
//		z = M \ r;
//	z_tld = M' \ r_tld;
//		rho = (z'*r_tld );
//		if (rho == 0.0),
//			break
//			end
//
//			if (iter > 1), % direction vectors
//				beta = rho / rho_1;
//	p = z + beta*p;
//	p_tld = z_tld + beta*p_tld;
//			else
//				p = z;
//	p_tld = z_tld;
//	end
//
//		q = A*p;                            % compute residual pair
//		q_tld = A'*p_tld;
//		alpha = rho / (p_tld'*q );
//
//		x = x + alpha*p;                    % update approximation
//		r = r - alpha*q;
//	r_tld = r_tld - alpha*q_tld;
//
//	error = norm(r) / bnrm2;          % check convergence
//		if (error <= tol), break, end
//
//			rho_1 = rho;
//
//	end
//
//		if (error <= tol), % converged
//			flag = 0;
//	elseif(rho == 0.0), % breakdown
//		flag = -1;
//		else
//			flag = 1;                           % no convergence
//			end
//
//}