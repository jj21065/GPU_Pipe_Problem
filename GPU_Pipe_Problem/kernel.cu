#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include<windows.h>

#define N 29
float rs0 = 1;
float r01 = 1;
float r16 = 5;
float r12 = 1;
float r03 = 5;
float r34 = 3;
float r24 = 1;
float r25 = 1;
float r56 = 3;
float r3e = 1;
float r47 = 1;

float r57 = 1;
float r78 = 2;
float r89 = 3;
float r69 = 3;

float r8a = 1;
float reb = 2;
float rdg = 5;
float ref = 5;

float rab = 3;
float rbc = 2;
float r9g = 1;
float rcf = 2;
float rad = 2;
float rdc = 1;
float rgh = 1;
float rfh = 2;
float r9EA = 1;
float rhEB = 1;

float Qs0 = Qs0 = sqrt(10.0 / rs0);
float Q01 = Qs0;
float Q16 = Qs0;
float Q12 = 0;
float Q03 = 0;
float Q34 = 0;
float Q24 = 0;
float Q25 = 0;
float Q56 = 0;
float Q3e = 0;
float Q47 = 0;

float Q57 = 0;
float Q78 = 0;
float Q89 = 0;
float Q69 = Qs0;

float Q8a = 0;
float Qeb = 0;
float Qdg = 0;
float Qef = 0;

float Qab = 0;
float Qbc = 0;
float Q9g = 0;
float Qcf = 0;

float Qad = 0;
float Qdc = 0;
float Qgh = 0;
float Qfh = 0;


float Q9EA = Qs0;
float QhEB = 0;


void MatrixVectorProduct(float *a, float*p, float *ap, int n);

float Dot(float *v1, float *v2, int n);

void Sum_Store(float *store, float *x, float scalar, float*v, int n);

void conjgrad(float*A, float*x, float*b);

void Newton_Raphson_method(float *x, float*r);

void Compute_R(float*x, float*r, float *R);

void Compute_J(float*x, float*r, int n);

void Compute_invJ_mul_R(float*x, float*r, float*R, float*invJ_R);

int main()
{
	float* Q = new float[N];
	float* r = new float[N];
	// pipe parameter 
	Q[0] = sqrt(10.0 / rs0);
	Q[1] = 1;
	Q[2] = 1;
	Q[3] = 1;
	Q[4] = 1;
	Q[5] = 1;
	Q[6] = 1;
	Q[7] = 1;
	Q[8] = 1;
	Q[9] = 1;
	Q[10] = 1;
	Q[11] = 1;
	Q[12] = 1;
	Q[13] = 1;
	Q[14] = 1;
	Q[15] = 1;
	Q[16] = 1;
	Q[17] = 1;
	Q[18] = 1;
	Q[19] = 1;
	Q[20] = 1;
	Q[21] = 1;
	Q[22] = 1;
	Q[23] = 1;
	Q[24] = 1;
	Q[25] = 1;
	Q[26] = 1;
	Q[27] = 1;
	Q[28] = 1;

	r[0] = 1;
	r[1] = 1;
	r[2] = 5;
	r[3] = 1;
	r[4] = 5;
	r[5] = 3;
	r[6] = 1;
	r[7] = 1;
	r[8] = 3;
	r[9] = 1;
	r[10] = 1;
	r[11] = 1;
	r[12] = 2;
	r[13] = 3;
	r[14] = 3;
	r[15] = 1;
	r[16] = 2;
	r[17] = 5;
	r[18] = 5;
	r[19] = 3;
	r[20] = 2;
	r[21] = 1;
	r[22] = 2;
	r[23] = 2;
	r[24] = 1;
	r[25] = 1;
	r[26] = 2;
	r[27] = 1;
	r[28] = 1;


	// initial conditions 

	int iter_no = 10;
	int i;


	printf("inital Qs0 = %g\n", Q[0]);
	for (i = 0; i < iter_no; i++)
	{
		//	Hardy_Cross_method();


		Newton_Raphson_method(Q, r);
	}

	printf("final Qs0 = %g\n", Q[0]);
	printf("final Q9-EA = %g\n", Q[27]);
	printf("final Qh-EB = %g\n", Q[28]);

	free(Q);
	free(r);
	system("pause");
	return 0;
}

void Hardy_Cross_method()
{
	float R[11] = { 0 };

	/// calculate adjust flow Q
	if (2 * (r01*abs(Q01) + r12*abs(Q12) + r24*abs(Q24) + r34*abs(Q34) + r03*abs(Q03)) > 0)
		R[0] = -(r01*Q01*abs(Q01) + r12*Q12*abs(Q12) + r24*Q24*abs(Q24) - r34*Q34*abs(Q34) - r03*Q03*abs(Q03)) / (2 * (r01*abs(Q01) + r12*abs(Q12) + r24*abs(Q24) + r34*abs(Q34) + r03*abs(Q03)));

	if (2 * (r16*abs(Q16) + r56*abs(Q56) + r25*abs(Q25) + r12*abs(Q12)) > 0)
		R[1] = -(r16*Q16*abs(Q16) - r56*Q56*abs(Q56) - r25*Q25*abs(Q25) - r12*Q12*abs(Q12)) / (2 * (r16*abs(Q16) + r56*abs(Q56) + r25*abs(Q25) + r12*abs(Q12)));

	if (2 * (r34*abs(Q34) + r47*abs(Q47) + r78*abs(Q78) + r8a*abs(Q8a) + rab*abs(Qab) + reb*abs(Qeb) + r3e*abs(Q3e)) > 0)
		R[2] = -(r34*Q34*abs(Q34) + r47*Q47*abs(Q47) + r78*Q78*abs(Q78) + r8a*Q8a*abs(Q8a) + rab*Qab*abs(Qab) - reb*Qeb*abs(Qeb) - r3e*Q3e*abs(Q3e)) / (2 * (r34*abs(Q34) + r47*abs(Q47) + r78*abs(Q78) + r8a*abs(Q8a) + rab*abs(Qab) + reb*abs(Qeb) + r3e*abs(Q3e)));

	if (2 * (r56*abs(Q56) + r69*abs(Q69) + r89*abs(Q89) + r78*abs(Q78) + r57*abs(Q57)) > 0)
		R[3] = -(r56*Q56*abs(Q56) + r69*Q69*abs(Q69) - r89*Q89*abs(Q89) - r78*Q78*abs(Q78) - r57*Q57*abs(Q57)) / (2 * (r56*abs(Q56) + r69*abs(Q69) + r89*abs(Q89) + r78*abs(Q78) + r57*abs(Q57)));

	if (2 * (r89*abs(Q89) + r9g*abs(Q9g) + rdg*abs(Qdg) + rad*abs(Qad) + r8a*abs(Q8a)) > 0)
		R[4] = -(r89*Q89*abs(Q89) - r9g*Q9g*abs(Q9g) - rdg*Qdg*abs(Qdg) - rad*Qad*abs(Qad) - r8a*Q8a*abs(Q8a)) / (2 * (r89*abs(Q89) + r9g*abs(Q9g) + rdg*abs(Qdg) + rad*abs(Qad) + r8a*abs(Q8a)));

	if (2 * (rad*abs(Qad) + rdc*abs(Qdc) + rbc*abs(Qbc) + rab*abs(Qab)) > 0)
		R[5] = -(rad*Qad*abs(Qad) + rdc*Qdc*abs(Qdc) - rbc*Qbc*abs(Qbc) - rab*Qab*abs(Qab)) / (2 * (rad*abs(Qad) + rdc*abs(Qdc) + rbc*abs(Qbc) + rab*abs(Qab)));

	if (2 * (reb*abs(Qeb) + rbc*abs(Qbc) + rcf*abs(Qcf) + ref*abs(Qef)) > 0)
		R[6] = -(reb*Qeb*abs(Qeb) + rbc*Qbc*abs(Qbc) + rcf*Qcf*abs(Qcf) - ref*Qef*abs(Qef)) / (2 * (reb*abs(Qeb) + rbc*abs(Qbc) + rcf*abs(Qcf) + ref*abs(Qef)));

	if (2 * (rdg*abs(Qdg) + rgh*abs(Qgh) + rfh*abs(Qfh) + rcf*abs(Qcf) + rdc*abs(Qdc)) > 0)
		R[7] = -(rdg*Qdg*abs(Qdg) + rgh*Qgh*abs(Qgh) - rfh*Qfh*abs(Qfh) - rcf*Qcf*abs(Qcf) - rdc*Qdc*abs(Qdc)) / (2 * (rdg*abs(Qdg) + rgh*abs(Qgh) + rfh*abs(Qfh) + rcf*abs(Qcf) + rdc*abs(Qdc)));

	if (2 * (r25*abs(Q25) + r57*abs(Q57) + r47*abs(Q47) + r24*abs(Q24)) > 0)
		R[8] = -(r25*Q25*abs(Q25) + r57*Q57*abs(Q57) - r47*Q47*abs(Q47) - r24*Q24*abs(Q24)) / (2 * (r25*abs(Q25) + r57*abs(Q57) + r47*abs(Q47) + r24*abs(Q24)));

	if (2 * (rs0*abs(Qs0) + r01*abs(Q01) + r16*abs(Q16) + r69*abs(Q69) + r9EA*abs(Q9EA)) > 0)
		R[9] = -(rs0*Qs0*abs(Qs0) + r01*Q01*abs(Q01) + r16*Q16*abs(Q16) + r69*Q69*abs(Q69) + r9EA*Q9EA*abs(Q9EA) - 10) / (2 * (rs0*abs(Qs0) + r01*abs(Q01) + r16*abs(Q16) + r69*abs(Q69) + r9EA*abs(Q9EA)));

	if (2 * (rs0*abs(Qs0) + r03*abs(Q03) + r3e*abs(Q3e) + ref*abs(Qef) + rfh*abs(Qfh) + rhEB*abs(QhEB)) > 0)
		R[10] = -(rs0*Qs0*abs(Qs0) + r03*Q03*abs(Q03) + r3e*Q3e*abs(Q3e) + ref*Qef*abs(Qef) + rfh*Qfh*abs(Qfh) + rhEB*QhEB*abs(QhEB) - 10) / (2 * (rs0*abs(Qs0) + r03*abs(Q03) + r3e*abs(Q3e) + ref*abs(Qef) + rfh*abs(Qfh) + rhEB*abs(QhEB)));

	/// add the adjust flow to each Q pipes
	Q01 = Q01 + R[0] + R[9];
	Q12 = Q12 + R[0] - R[1];
	Q24 = Q24 + R[0] - R[8];
	Q34 = Q34 - R[0] + R[2];
	Q03 = Q03 - R[0] + R[10];
	Q16 = Q16 + R[1] + R[9];
	Q56 = Q56 - R[1] + R[3];
	Q25 = Q25 - R[1] + R[8];
	Q47 = Q47 - R[8] + R[2];
	Q78 = Q78 + R[2] - R[3];
	Q8a = Q8a + R[2] - R[4];

	Q57 = Q57 - R[3] + R[8];
	Q89 = Q89 - R[3] + R[4];
	Q69 = Q69 + R[3] + R[9];
	Qad = Qad - R[4] + R[5];
	Q9g = Q9g - R[4];
	Qdg = Qdg - R[4] + R[7];
	Qdc = Qdc + R[5] - R[7];
	Qgh = Qgh + R[7];
	Qfh = Qfh - R[7] + R[10];
	QhEB = QhEB + R[10];
	Q9EA = Q9EA + R[9];
	Qab = Qab + R[2] - R[5];
	Qeb = Qeb - R[2] + R[6];
	Q3e = Q3e - R[2] + R[10];
	Qef = Qef - R[6] + R[10];
	Qbc = Qbc - R[5] + R[6];
	Qcf = Qcf + R[6] - R[7];
	Qs0 = Qs0 + R[10] + R[9];
}

void Newton_Raphson_method(float *x, float*r)
{

	/// Computet R 
	float invJR[N] = { 0 };
	float R[N] = { 0 };
	Compute_R(x, r, R);
	/// Compute J 
	Compute_invJ_mul_R(x, r, R, invJR);

	for (int i = 0; i < N; i++)
	{
		x[i] = x[i] - invJR[i];
	}
	// x = x - inv(J')*R'
}

void Compute_R(float*x, float*r, float *R)
{

	R[0] = (r01*x[1] * abs(x[1]) + r12*x[3] * abs(x[3]) + r24*x[6] * abs(x[6]) - r34*x[5] * abs(x[5]) - r03*x[4] * abs(x[4]));

	R[1] = (r16*x[2] * abs(x[2]) - r56*x[8] * abs(x[8]) - r25*x[7] * abs(x[7]) - r12*x[3] * abs(x[3]));

	R[2] = (r34*x[5] * abs(x[5]) + r47*x[10] * abs(x[10]) + r78*x[12] * abs(x[12]) + r8a*x[15] * abs(x[15]) + rab*x[19] * abs(x[19]) - reb*x[16] * abs(x[16]) - r3e*x[9] * abs(x[9]));

	R[3] = (r56*x[8] * abs(x[8]) + r69*x[14] * abs(x[14]) - r89*x[13] * abs(x[13]) - r78*x[12] * abs(x[12]) - r57*x[11] * abs(x[11]));

	R[4] = (r89*x[13] * abs(x[13]) - r9g*x[21] * abs(x[21]) - rdg*x[17] * abs(x[17]) - rad*x[23] * abs(x[23]) - r8a*x[15] * abs(x[15]));

	R[5] = (rad*x[23] * abs(x[23]) + rdc*x[24] * abs(x[24]) - rbc*x[20] * abs(x[20]) - rab*x[19] * abs(x[19]));

	R[6] = (reb*x[16] * abs(x[16]) + rbc*x[20] * abs(x[20]) + rcf*x[22] * abs(x[22]) - ref*x[18] * abs(x[18]));

	R[7] = (rdg*x[17] * abs(x[17]) + rgh*x[25] * abs(x[25]) - rfh*x[26] * abs(x[26]) - rcf*x[22] * abs(x[22]) - rdc*x[24] * abs(x[24]));

	R[8] = (r25*x[7] * abs(x[7]) + r57*x[11] * abs(x[11]) - r47*x[10] * abs(x[10]) - r24*x[6] * abs(x[6]));

	R[9] = (rs0*x[0] * abs(x[0]) + r01*x[1] * abs(x[1]) + r16*x[2] * abs(x[2]) + r69*x[14] * abs(x[14]) + r9EA*x[27] * abs(x[27]) - 10);

	R[10] = (rs0*x[0] * abs(x[0]) + r03*x[4] * abs(x[4]) + r3e*x[9] * abs(x[9]) + ref*x[18] * abs(x[18]) + rfh*x[26] * abs(x[26]) + rhEB*x[28] * abs(x[28]) - 10);

	R[11] = x[1] + x[4] - x[0];
	
	R[12] = x[2] + x[3] - x[1];
	
	R[14] = x[6] + x[7] - x[3];

	R[15] = x[5] + x[9] - x[4];

	R[16] = x[5] + x[6] - x[10];

	R[17] = x[10] + x[11] - x[12];

	R[18] = x[13] + x[15] - x[12];

	R[19] = x[8] + x[11] - x[7];

	R[20] = x[2] + x[8] - x[14];

	R[21] = x[14] + x[13] + x[21] - x[27];

	R[22] = x[19] + x[23] - x[15];

	R[23] = x[16] + x[19] - x[20];

	R[24] = x[16] + x[19] - x[9];

	R[25] = x[20] + x[24] - x[22];

	R[26] = x[22] + x[18] - x[26];

	R[27] = x[21] + x[25] - x[17];

	R[28] = x[25] + x[26] - x[28];
	

}

void Compute_invJ_mul_R(float*x, float*r,float*R,float*invJ_R)
{

	/// Conjugate gradient method 
	float A[N*N] = { 0 };

	A[0] = 0; A[1] = 2 * r[1] * x[1];  A[3] = 2 * r[3] * x[3]; A[4] = -2 * r[4] * x[4]; A[5] = -2 * r[5] * x[5]; A[6] = 2 * r[6] * x[6];
	
	A[29] = 0; A[30] = 2 * r[2] * x[2]; A[31] = -2 * r[3] * x[3]; A[36] = 2 * r[7] * x[7]; A[37] = 2 * r[8] * x[8];
	
	A[58] = 0; A[63] = 2 * r[5] * x[5]; A[67] = -2 * r[9] * x[9]; A[68] = 2 * r[10] * x[10]; A[70] = 2 * r[12] * x[12]; A[73] = 2 * r[15] * x[15];A[74] = -2*r[16]*x[16]; A[77] = 2 * r[19] * x[19];
	
	A[87] = 0; A[95] = 2 * r[8] * x[8]; A[98] = -2 * r[11] * x[11]; A[99] = -2 * r[12] * x[12]; A[100] = -2 * r[13] * x[13]; A[101] = 2 * r[14] * x[14];
	
	A[116] = 0; A[129] = 2 * r[13] * x[13]; A[131] = -2 * r[15] * x[15]; A[133] = -2 * r[17] * x[17]; A[137] = 2 * r[21] * x[21]; A[139] = 2 * r[23] * x[23];
	
	A[145] = 0; A[164] = -2 * r[19] * x[19]; A[165] = -2 * r[20] * x[20]; A[168] = 2 * r[23] * x[23]; A[169] = 2 * r[24] * x[24];

	A[174] = 0; A[190] = 2 * r[16] * x[16]; A[192] = -2 * r[18] * x[18]; A[194] = 2 * r[20] * x[20]; A[196] = 2 * r[22] * x[22];

	A[203] = 0; A[220] = 2 * r[17] * x[17]; A[225] = -2 * r[22] * x[22]; A[227] = -2 * r[24] * x[24]; A[228] = 2 * r[25] * x[25]; A[229] = -2 * r[26] * x[26];

	A[232] = 0; A[238] = -2 * r[6] * x[6]; A[239] = 2 * r[7] * x[7]; A[242] = -2 * r[10] * x[10]; A[243] = 2 * r[11] * x[11];
	
	A[261] = 2 * r[0] * x[0]; A[262] = 2 * r[1] * x[1]; A[263] = 2 * r[2] * x[2]; A[275] = 2 * r[14] * x[14]; A[288] = 2 * r[27] * x[27];

	A[290] = 2 * r[0] * x[0]; A[294] = 2 * r[4] * x[4]; A[299] = 2 * r[9] * x[9]; A[308] = 2 * r[18] * x[18]; A[316] = 2 * r[26] * x[26]; A[318] = 2 * r[28] * x[28];

	A[319] = -1; x[320] = 1; x[323] = 1;

	A[348] = 0; A[349] = -1; A[350] = 1; A[351] = 1;

	A[377] = 0; A[380] = -1; A[383] = 1; A[384] = 1;

	A[406] = 0; A[410] = -1; A[411] = 1; A[415] = 1;

	A[435] = 0; A[440] = 1; A[441] = 1; A[445] = -1; 

	A[464] = 0; A[474] = 1; A[475] = 1; A[476] = -1;

	A[493] = 0; A[503] = 1; A[504] = 1; A[505] = -1;

	A[522] = 0; A[534] = -1; A[535] = 1; A[537] = 1;

	A[551] = 0; A[558] = -1; A[559] = 1; A[562] = 1;

	A[580] = 0; A[582] = 1; A[588] = 1; A[594] = -1;

	A[609] = 0; A[622] = 1; A[623] = 1; A[630] = 1; A[636] = -1;

	A[638] = 0; A[653] = -1; A[657] = 1; A[661] = 1;

	A[667] = 0; A[683] = 1; A[686] = 1; A[687] = -1;

	A[696] = 0; A[705] = -1; A[712] = 1; A[715] = 1;

	A[725] = 0; A[745] = 1; A[749] = 1; A[747] = -1;

	A[754] = 0; A[772] = 1; A[776] = 1; A[780] = -1;

	A[783] = 0; A[800] = -1; A[804] = 1; A[808] = 1;
	
	A[812] = 0; A[837] = 1; A[838] = 1; A[840] = -1;

	//FILE*pfile;
	//pfile = fopen("output.txt", "w");

	//for (int i = 0; i < N; i++)
	//{
	//	for (int j = 0; j < N; j++)
	//	{
	//		fprintf(pfile, "%g\t", A[i*N + j]);
	//	}
	//	fprintf(pfile, "\n");
	//}
	//fclose(pfile);
	conjgrad(A, invJ_R, R);

}

void MatrixVectorProduct(float *a, float*p, float *ap, int n)
{
	int i = 0;
	int j = 0;

	for (i = 0; i < n; i++){
		ap[i] = 0.0;
		for (j = 0; j < n; j++){
			ap[i] = ap[i] + a[i*n + j] * p[j];
		}
	}
}

float Dot(float *v1, float *v2, int n)
{
	int i;
	float ans = 0.0;
	for (i = 0; i < n; i++){
		ans += v1[i] * v2[i];

	}
	return ans;
}

void Sum_Store(float *store, float *x, float scalar, float*v, int n)
{
	int i = 0;
	for (i = 0; i < n; i++){
		store[i] = x[i] + scalar * v[i];
	}
}

void conjgrad(float*A, float*x, float*b)
{
	float ap[N];
	float p[N];
	float rr[N];
	float rsold = 0;
	float rsnew = 0;
	int i = 0;
	MatrixVectorProduct(A, x, ap, N);
	for (i = 0; i < N; i++){
		rr[i] = b[i] - ap[i];
		p[i] = rr[i];
	}
	rsold = Dot(rr, rr, N);

	for (i = 0; i < N; i++){
		MatrixVectorProduct(A, p, ap, N);
	/*	for (int j = 0; j < N*N; j++){
			printf("ap[%d] = %g\n", j, ap[j]);
		}*/
		float tmpap = Dot(ap, p, N);
		float alpha = rsold / (tmpap);
		Sum_Store(x, x, alpha, p, N);
		Sum_Store(rr, rr, -alpha, ap, N);
		rsnew = Dot(rr, rr, N);
		if (sqrt(rsnew) < 1e-10)
			break;
		Sum_Store(p, rr, (rsnew / rsold), p, N);
		rsold = rsnew;

	}
// 	for (i = 0; i < N; i++)
// 		printf("x[%d] = %g\n", i, x[i]);


}