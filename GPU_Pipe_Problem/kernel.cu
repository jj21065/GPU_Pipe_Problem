
#include "hip/hip_runtime.h"


#include <stdio.h>

float mypow(float, int);
float myabs(float a)
{
	if (a < 0)
		return -a;
	else
		return a;
}
int main()
{
	// pipe parameter
	float r12 = 5;
	float r13 = 1;
	float r23 = 10;
	float r24 = 1;
	float r34 = 5;
	// initial conditions
	float Q12 = 0;
	float Q13 = 10;
	float Q23 = -3;
	float Q24 = Q12 - Q23;
	float Q34 = Q13 + Q23;

	int n = 2;
	int iter_no = 500;
	int i;
	//r23 = (r24*Q24*Q24-r34*Q34*Q34)/Q23/Q23;
	//r23 = (r12*Q12*Q12-r13*Q13*Q13)/Q23/Q23;
	int c12 = 1, c13 = 1, c23 = 1, c24 = 1, c34 = 1;

	for (i = 0; i < iter_no; i++)
	{
		//Control the direction sign
		c12 = (Q12 >= 0) ? 1 : -1;
		c13 = (Q13 >= 0) ? 1 : -1;
		c23 = (Q23 >= 0) ? 1 : -1;
		c24 = (Q24 >= 0) ? 1 : -1;
		c34 = (Q34 >= 0) ? 1 : -1;
		float dQ_2 = -(r24*c24*Q24*Q24 - c34*r34*Q34*Q34 - c23*r23*Q23*Q23) / (2 * (r24*myabs(Q24) + r34*myabs(Q34) + r23*myabs(Q23)));
		Q12 = Q12 + dQ_2;
		Q13 = Q13 - dQ_2;
		Q24 = Q24 + dQ_2;
		Q34 = Q34 - dQ_2;
		//calculate the new r23 from the last Q
		r23 = myabs((-r12*c12*Q12*Q12 + c13*r13*Q13*Q13) / (Q23*Q23*c23));
	}


	printf("\n\n");
	printf("Q12 = %g \n", Q12);
	printf("Q13 = %g \n", Q13);
	printf("Q23 = %g \n", Q23);
	printf("Q24 = %g \n", Q24);
	printf("Q34 = %g \n\n", Q34);
	printf("r23 = %g \n", r23);
	system("pause");
	return 0;
}
float mypow(float value, int n)
{
	int i;
	if (n == 0)
		return 1;
	for (i = 0; i < n - 1; i++)
	{
		value = value*value;
	}
	return value;
}
