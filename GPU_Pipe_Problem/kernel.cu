
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
float myabs(float);
float mypow(float, int);
int main()
{
	// pipe parameter 
	float r01 = 1;
	float r16 = 5;
	float r12 = 1;
	float r03 = 5;
	float r34 = 3;

	float r24 = 1;
	float r25 = 1;
	float r56 = 3;
	float r3e = 1;
	float r47 = 1;

	float r57 = 1;
	float r78 = 2;
	float r89 = 3;
	float r69 = 3;

	float r8a = 1;
	float reb = 2;
	float rdg = 5;
	float ref = 5;

	float rab = 3;
	float rbc = 2;
	float r9g = 1;
	float rcf = 2;

	float rad = 2;
	float rdc = 1;
	float rgh = 1;
	float rfh = 2;

	float rs0 = 1;
	float r9EA = 1;
	float rhEB = 1;


	float Qs0 = Qs0 = sqrt(10.0 / rs0);
	float Q01 = Qs0;
	float Q16 = Qs0;
	float Q12 = 0;
	float Q03 = 0;
	float Q34 = 0;
	float Q24 = 0;
	float Q25 = 0;
	float Q56 = 0;
	float Q3e = 0;
	float Q47 = 0;

	float Q57 = 0;
	float Q78 = 0;
	float Q89 = 0;
	float Q69 = Qs0;

	float Q8a = 0;
	float Qeb = 0;
	float Qdg = 0;
	float Qef = 0;

	float Qab = 0;
	float Qbc = 0;
	float Q9g = 0;
	float Qcf = 0;

	float Qad = 0;
	float Qdc = 0;
	float Qgh = 0;
	float Qfh = 0;


	float Q9EA = Qs0;
	float QhEB = 0;


	float R[11] = { 0 };

	// initial conditions 


	int n = 2;
	int iter_no = 5000;
	int i;
	clock_t t1 = clock();
	printf("inital Qs0 = %g\n", Qs0);
	for (i = 0; i < iter_no; i++)
	{
		/// calculate adjust flow Q
		if (2 * (r01*abs(Q01) + r12*abs(Q12) + r24*abs(Q24) + r34*abs(Q34) + r03*abs(Q03)) > 0)
			R[0] = -(r01*Q01*abs(Q01) + r12*Q12*abs(Q12) + r24*Q24*abs(Q24) - r34*Q34*abs(Q34) - r03*Q03*abs(Q03)) / (2 * (r01*abs(Q01) + r12*abs(Q12) + r24*abs(Q24) + r34*abs(Q34) + r03*abs(Q03)));
		
		if (2 * (r16*abs(Q16) + r56*abs(Q56) + r25*abs(Q25) + r12*abs(Q12)) > 0)
			R[1] = -(r16*Q16*abs(Q16) - r56*Q56*abs(Q56) - r25*Q25*abs(Q25) - r12*Q12*abs(Q12)) / (2 * (r16*abs(Q16) + r56*abs(Q56) + r25*abs(Q25) + r12*abs(Q12)));
		
		if (2 * (r34*abs(Q34) + r47*abs(Q47) + r78*abs(Q78) + r8a*abs(Q8a) + rab*abs(Qab) + reb*abs(Qeb) + r3e*abs(Q3e)) > 0)
			R[2] = -(r34*Q34*abs(Q34) + r47*Q47*abs(Q47) + r78*Q78*abs(Q78) + r8a*Q8a*abs(Q8a) + rab*Qab*abs(Qab) - reb*Qeb*abs(Qeb) - r3e*Q3e*abs(Q3e)) / (2 * (r34*abs(Q34) + r47*abs(Q47) + r78*abs(Q78) + r8a*abs(Q8a) + rab*abs(Qab) + reb*abs(Qeb) + r3e*abs(Q3e)));
		
		if (2 * (r56*abs(Q56) + r69*abs(Q69) + r89*abs(Q89) + r78*abs(Q78) + r57*abs(Q57)) > 0)
			R[3] = -(r56*Q56*abs(Q56) + r69*Q69*abs(Q69) - r89*Q89*abs(Q89) - r78*Q78*abs(Q78) - r57*Q57*abs(Q57)) / (2 * (r56*abs(Q56) + r69*abs(Q69) + r89*abs(Q89) + r78*abs(Q78) + r57*abs(Q57)));
		
		if (2 * (r89*abs(Q89) + r9g*abs(Q9g) + rdg*abs(Qdg) + rad*abs(Qad) + r8a*abs(Q8a)) > 0)
			R[4] = -(r89*Q89*abs(Q89) - r9g*Q9g*abs(Q9g) - rdg*Qdg*abs(Qdg) - rad*Qad*abs(Qad) - r8a*Q8a*abs(Q8a)) / (2 * (r89*abs(Q89) + r9g*abs(Q9g) + rdg*abs(Qdg) + rad*abs(Qad) + r8a*abs(Q8a)));
		
		if (2 * (rad*abs(Qad) + rdc*abs(Qdc) + rbc*abs(Qbc) + rab*abs(Qab)) > 0)
			R[5] = -(rad*Qad*abs(Qad) + rdc*Qdc*abs(Qdc) - rbc*Qbc*abs(Qbc) - rab*Qab*abs(Qab)) / (2 * (rad*abs(Qad) + rdc*abs(Qdc) + rbc*abs(Qbc) + rab*abs(Qab)));
		
		if (2 * (reb*abs(Qeb) + rbc*abs(Qbc) + rcf*abs(Qcf) + ref*abs(Qef)) > 0)
			R[6] = -(reb*Qeb*abs(Qeb) + rbc*Qbc*abs(Qbc) + rcf*Qcf*abs(Qcf) - ref*Qef*abs(Qef)) / (2 * (reb*abs(Qeb) + rbc*abs(Qbc) + rcf*abs(Qcf) + ref*abs(Qef)));
		
		if (2 * (rdg*abs(Qdg) + rgh*abs(Qgh) + rfh*abs(Qfh) + rcf*abs(Qcf) + rdc*abs(Qdc)) > 0)
			R[7] = -(rdg*Qdg*abs(Qdg) + rgh*Qgh*abs(Qgh) - rfh*Qfh*abs(Qfh) - rcf*Qcf*abs(Qcf) - rdc*Qdc*abs(Qdc)) / (2 * (rdg*abs(Qdg) + rgh*abs(Qgh) + rfh*abs(Qfh) + rcf*abs(Qcf) + rdc*abs(Qdc)));
		
		if (2 * (r25*abs(Q25) + r57*abs(Q57) + r47*abs(Q47) + r24*abs(Q24)) > 0)
			R[8] = -(r25*Q25*abs(Q25) + r57*Q57*abs(Q57) - r47*Q47*abs(Q47) - r24*Q24*abs(Q24)) / (2 * (r25*abs(Q25) + r57*abs(Q57) + r47*abs(Q47) + r24*abs(Q24)));
		
		if (2 * (rs0*abs(Qs0) + r01*abs(Q01) + r16*abs(Q16) + r69*abs(Q69) + r9EA*abs(Q9EA)) > 0)
			R[9] = -(rs0*Qs0*abs(Qs0) + r01*Q01*abs(Q01) + r16*Q16*abs(Q16) + r69*Q69*abs(Q69) + r9EA*Q9EA*abs(Q9EA) - 10) / (2 * (rs0*abs(Qs0) + r01*abs(Q01) + r16*abs(Q16) + r69*abs(Q69) + r9EA*abs(Q9EA)));
		
		if (2 * (rs0*abs(Qs0) + r03*abs(Q03) + r3e*abs(Q3e) + ref*abs(Qef) + rfh*abs(Qfh) + rhEB*abs(QhEB)) > 0)
			R[10] = -(rs0*Qs0*abs(Qs0) + r03*Q03*abs(Q03) + r3e*Q3e*abs(Q3e) + ref*Qef*abs(Qef) + rfh*Qfh*abs(Qfh) + rhEB*QhEB*abs(QhEB) -10) / (2 * (rs0*abs(Qs0) + r03*abs(Q03) + r3e*abs(Q3e) + ref*abs(Qef) + rfh*abs(Qfh) + rhEB*abs(QhEB)));

		/// add the adjust flow to each Q pipes
		Q01 = Q01 + R[0] + R[9];
		Q12 = Q12 + R[0] - R[1];
		Q24 = Q24 + R[0] - R[8];
		Q34 = Q34 - R[0] + R[2];
		Q03 = Q03 - R[0] + R[10];
		Q16 = Q16 + R[1] + R[9];
		Q56 = Q56 - R[1] + R[3];
		Q25 = Q25 - R[1] + R[8];
		Q47 = Q47 - R[8] + R[2];
		Q78 = Q78 + R[2] - R[3];
		Q8a = Q8a + R[2] - R[4];

		Q57 = Q57 - R[3] + R[8];
		Q89 = Q89 - R[3] + R[4];
		Q69 = Q69 + R[3] + R[9];
		Qad = Qad - R[4] + R[5];
		Q9g = Q9g - R[4];
		Qdg = Qdg - R[4] + R[7];
		Qdc = Qdc + R[5] - R[7];
		Qgh = Qgh + R[7];
		Qfh = Qfh - R[7] + R[10];
		QhEB = QhEB + R[10];
		Q9EA = Q9EA + R[9];
		Qab = Qab + R[2] - R[5];
		Qeb = Qeb - R[2] + R[6];
		Q3e = Q3e - R[2] + R[10];
		Qef = Qef - R[6] + R[10];
		Qbc = Qbc - R[5] + R[6];
		Qcf = Qcf + R[6] - R[7];
		Qs0 = Qs0 + R[10] + R[9];

	}
	clock_t t2 = clock();

	printf("time consume : %f", t2 - t1);
	//	printf("abs = %g\n",myabs(-2));
	printf("final Qs0 = %g\n", Qs0);
	printf("final Q9-EA = %g\n", Q9EA);
	printf("final Qh-EB = %g\n", QhEB);
	
	system("pause");
	return 0;
}

